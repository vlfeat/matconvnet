#include "hip/hip_runtime.h"
// @file vl_nnpool.cu
// @brief Pooling block MEX wrapper
// @author Andrea Vedaldi
// @author Karel Lenc

/*
Copyright (C) 2014-15 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "bits/mexutils.h"
#include "bits/datamex.hpp"
#include "bits/nnpooling.hpp"

#if ENABLE_GPU
#include "bits/datacu.hpp"
#endif

#include <assert.h>

/* option codes */
enum {
  opt_stride = 0,
  opt_pad,
  opt_method,
  opt_verbose,
  opt_cudnn,
  opt_no_cudnn,
} ;

/* options */
vlmxOption  options [] = {
  {"Stride",           1,   opt_stride            },
  {"Pad",              1,   opt_pad               },
  {"Method",           1,   opt_method            },
  {"Verbose",          0,   opt_verbose           },
  {"CUDNN",            0,   opt_cudnn             },
  {"NoCUDNN",          0,   opt_no_cudnn          },
  {0,                  0,   0                     }
} ;

/* ---------------------------------------------------------------- */
/*                                                          Context */
/* ---------------------------------------------------------------- */

vl::MexContext context ;

/*
 Resetting the context here resolves a crash when MATLAB quits and
 the ~Context function is implicitly called on unloading the MEX file.
 */
void atExit()
{
  context.clear() ;
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
  IN_DATA = 0, IN_SIZE, IN_DEROUTPUT, IN_END
} ;

enum {
  OUT_RESULT = 0, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
  int poolWidth ;
  int poolHeight ;
  int strideX = 1 ;
  int strideY = 1 ;
  int padLeft = 0 ;
  int padRight = 0 ;
  int padTop = 0 ;
  int padBottom = 0 ;
  vl::PoolingMethod method = vl::vlPoolingMax ;
  bool backMode = false ;

  int verbosity = 0 ;
  int opt ;
  int next = IN_END ;
  mxArray const *optarg ;

  /* -------------------------------------------------------------- */
  /*                                            Check the arguments */
  /* -------------------------------------------------------------- */

  mexAtExit(atExit) ;

  if (nin < 2) {
    mexErrMsgTxt("The arguments are less than two.") ;
  }

  if (nin > 2 && vlmxIsString(in[2],-1)) {
    next = 2 ;
    backMode = 0 ;
  } else {
    backMode = (nin >= 3) ;
  }

  while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
    switch (opt) {
      case opt_verbose :
        ++ verbosity ;
        break ;

      case opt_stride :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          mexErrMsgTxt("STRIDE is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            strideY = (int)mxGetPr(optarg)[0] ;
            strideX = strideY ;
            break ;
          case 2:
            strideY = (int)mxGetPr(optarg)[0] ;
            strideX = (int)mxGetPr(optarg)[1] ;
            break ;
          default:
            mexErrMsgTxt("STRIDE has neither one nor two elements.") ;
        }
        break ;

      case opt_pad :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          mexErrMsgTxt("PAD is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            padLeft = (int)mxGetPr(optarg)[0] ;
            padRight = padLeft ;
            padTop = padLeft ;
            padBottom = padLeft ;
            break ;
          case 4:
            padTop = (int)mxGetPr(optarg)[0] ;
            padBottom = (int)mxGetPr(optarg)[1] ;
            padLeft = (int)mxGetPr(optarg)[2] ;
            padRight = (int)mxGetPr(optarg)[3] ;
            break ;
          default:
            mexErrMsgTxt("PAD has neither one nor four elements.") ;
        }
        break;

      case opt_method :
        if (!vlmxIsString(optarg,-1)) {
           vlmxError(vlmxErrInvalidArgument, "METHOD is not a string.") ;
        }
        if (vlmxIsEqualToStringI(optarg, "max")) {
          method = vl::vlPoolingMax ;
        } else if (vlmxIsEqualToStringI(optarg, "avg")) {
          method = vl::vlPoolingAverage ;
        } else {
          vlmxError(vlmxErrInvalidArgument, "METHOD is not a supported method.") ;
        }
        break;

      case opt_no_cudnn :
#if ENABLE_CUDNN
        context.getCudaHelper().setCudnnEnabled(false) ;
#endif
        break ;

      case opt_cudnn :
#if ENABLE_CUDNN
        context.getCudaHelper().setCudnnEnabled(true) ;
#endif
        break ;

      default:
        break ;
    }
  }

  vl::MexTensor data(context) ;
  vl::MexTensor derOutput(context) ;

  data.init(in[IN_DATA]) ;
  data.reshape(4) ; // -> 4 dimensions

  if (backMode) {
    derOutput.init(in[IN_DEROUTPUT]) ;
    derOutput.reshape(4) ; // -> 4 dimensions
  }

  if (backMode && ! vl::areCompatible(data, derOutput)) {
    mexErrMsgTxt("DATA and DEROUTPUT do not have compatible formats.") ;
  }

  if (!vlmxIsPlainMatrix(in[IN_SIZE],-1,-1)) {
    mexErrMsgTxt("SIZE is not a plain matrix.") ;
  }
  switch (mxGetNumberOfElements(in[IN_SIZE])) {
    case 1:
      poolHeight = mxGetPr(in[IN_SIZE])[0] ;
      poolWidth = poolHeight ;
      break ;
    case 2:
      poolHeight = mxGetPr(in[IN_SIZE])[0] ;
      poolWidth = mxGetPr(in[IN_SIZE])[1] ;
      break ;
    default:
      mexErrMsgTxt("SIZE has neither one nor two elements.") ;
  }

  /* Basic compatibility of Shape */
  if (strideX < 1 || strideY < 1) {
    mexErrMsgTxt("At least one element of STRIDE is smaller than one.") ;
  }
  if (poolHeight == 0 || poolWidth == 0) {
    mexErrMsgTxt("A dimension of the pooling SIZE is void.") ;
  }
  if (data.getHeight() + (padTop+padBottom) < poolHeight ||
      data.getWidth() + (padLeft+padRight) < poolWidth) {
    mexErrMsgTxt("The pooling window is larger than the DATA (including padding).") ;
  }
  if (padLeft < 0 ||
      padRight < 0 ||
      padTop < 0 ||
      padBottom < 0) {
    mexErrMsgTxt("An element of PAD is negative.") ;
  }
  if (padLeft >= poolWidth ||
      padRight >= poolWidth ||
      padTop >= poolHeight  ||
      padBottom >= poolHeight) {
    mexErrMsgTxt("A padding value is larger or equal to the size of the pooling window.") ;
  }

  /* Get the output Shape */
  vl::TensorShape outputShape((data.getHeight() + (padTop+padBottom) - poolHeight)/strideY + 1,
                              (data.getWidth()  + (padLeft+padRight) - poolWidth)/strideX + 1,
                              data.getDepth(),
                              data.getSize()) ;

  if (backMode && (derOutput != outputShape)) {
    mexErrMsgTxt("DEROUTPUT dimensions are incompatible with X and POOL.") ;
  }

  /* Create output buffers */
  vl::Device deviceType = data.getDeviceType() ;
  vl::Type dataType = data.getDataType() ;
  vl::MexTensor output(context) ;
  vl::MexTensor derData(context) ;

  if (!backMode) {
    output.initWithZeros(deviceType, dataType, outputShape) ;
  } else {
    derData.initWithZeros(deviceType, dataType, data.getShape()) ;
  }

  if (verbosity > 0) {
    mexPrintf("vl_nnpool: %s; %s", backMode?"backward":"forward", (data.getDeviceType()==vl::GPU) ? "GPU" : "CPU") ;
    if (data.getDeviceType() == vl::GPU) {
#if ENABLE_CUDNN
      mexPrintf("; %s\n", context.getCudaHelper().getCudnnEnabled() ? "cuDNN" : "MatConvNet") ;
#else
      mexPrintf("; MatConvNet\n") ;
#endif
    } else {
      mexPrintf("; MatConvNet\n") ;
    }
    mexPrintf("vl_nnpool: stride: [%d %d], pad: [%d %d %d %d]\n",
              strideY, strideX,
              padTop, padBottom, padLeft, padRight) ;
    vl::print("vl_nnpool: data: ", data) ;
    mexPrintf("vl_nnpool: pooling: %d x %d\n", poolHeight, poolWidth);
    mexPrintf("vl_nnpool: method: %s\n", (method == vl::vlPoolingMax) ? "max" : "avg") ;
    if (backMode) {
      vl::print("vl_nnpool: derOutput: ", derOutput) ;
      vl::print("vl_nnpool: derData: ", derData) ;
    } else {
      vl::print("vl_nnpool: output: ", output) ;
    }
  }

  /* -------------------------------------------------------------- */
  /*                                                    Do the work */
  /* -------------------------------------------------------------- */

  vl::Error error ;
  if (!backMode) {
    error = vl::nnpooling_forward(context,
                                  output, data,
                                  method,
                                  poolHeight, poolWidth,
                                  strideY, strideX,
                                  padTop, padBottom, padLeft, padRight) ;
  } else {
    error = vl::nnpooling_backward(context,
                                   derData, data, derOutput,
                                   method,
                                   poolHeight, poolWidth,
                                   strideY, strideX,
                                   padTop, padBottom, padLeft, padRight) ;
  }

  /* -------------------------------------------------------------- */
  /*                                                         Finish */
  /* -------------------------------------------------------------- */

  if (error != vl::vlSuccess) {
    mexErrMsgTxt(context.getLastErrorMessage().c_str()) ;
  }
  if (backMode) {
    out[OUT_RESULT] = derData.relinquish() ;
  } else {
    out[OUT_RESULT] = output.relinquish() ;
  }
}
