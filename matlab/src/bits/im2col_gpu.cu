#include "hip/hip_runtime.h"
/** @file im2col.cu
 ** @brief Image to columns and back (GPU)
 ** @author Andrea Vedaldi
 **/

/*
Copyright (C) 2014 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "im2col.cpp"
#include "gpu.hpp"

/* ---------------------------------------------------------------- */
/*                                                     im2col (GPU) */
/* ---------------------------------------------------------------- */

template <typename T>
__global__ void
im2col_gpu_kernel(T* stacked,
                  T const* data,
                  const int numPatchesX,
                  const int numPatchesY,
                  const int numPatchSlices,
                  const int width,
                  const int height,
                  const int windowWidth,
                  const int windowHeight,
                  const int strideX,
                  const int strideY,
                  const int padLeft,
                  const int padTop)
{
  /* each kernel copies the pixels in an image patch for one channel */
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < numPatchSlices) {
    /* 
      get the patch slice (x,y,z) to copy
     */
    int x = index ;
    int y = x / numPatchesX ;
    int z = y / numPatchesY ;
    x %= numPatchesX ;
    y %= numPatchesY ;

    /* 
     pick the top-left corer of the patch slice in the input image
     */
    int x_data = x * strideX - padLeft ;
    int y_data = y * strideY - padTop ;
    data += (z * height + y_data) * width + x_data ;

    /* 
     pick the column of the stacked image which contains this patch,
     and move down along the column at the beginning of the patch slice
     */
    int patchSliceOffset = (windowWidth*windowHeight) * z ;
    stacked += (numPatchesY * patchSliceOffset + y) * numPatchesX + x ;

    /*
     copy the patch slice
     */
    for (int v = 0 ; v < windowHeight ; ++v) {
      for (int u = 0 ; u < windowWidth ; ++u) {
        if (y_data + v >= 0 &&
            y_data + v < height &&
            x_data + u >= 0 &&
            x_data + u < width) {
          *stacked = data[v * width + u] ;
        } else {
          *stacked = 0 ;
        }
        stacked += (numPatchesX*numPatchesY) ;
      }
    }
  }
}

template <typename T>
void im2col_gpu(T* stacked,
                T const* data,
                size_t width,
                size_t height,
                size_t depth,
                size_t windowWidth,
                size_t windowHeight,
                size_t strideX,
                size_t strideY,
                size_t padLeft,
                size_t padRight,
                size_t padTop,
                size_t padBottom)
{
  int numPatchesX = (width + (padLeft + padRight) - windowWidth)/strideX + 1 ;
  int numPatchesY = (height + (padTop + padBottom) - windowHeight)/strideY + 1 ;
  int numPatchSlices = numPatchesX * numPatchesY * depth ;

  /*
   Each kernel copies a feature dimension of a patch.
   */
  im2col_gpu_kernel<T>
  <<< divideUpwards(numPatchSlices, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (stacked,
   data,
   numPatchesX,
   numPatchesY,
   numPatchSlices,
   width, height,
   windowWidth, windowHeight,
   strideX, strideY,
   padLeft, padTop) ;

  if (hipPeekAtLastError() != hipSuccess) {
    std::cout
    <<"im2col: CUDA kernel error ("
    <<hipGetErrorString(hipPeekAtLastError())
    <<")"<<std::endl ;
  }
}

// Explicit instantiation
template void im2col_gpu<float>(float* stacked,
                                float const* data,
                                size_t width,
                                size_t height,
                                size_t depth,
                                size_t windowWidth,
                                size_t windowHeight,
                                size_t strideX,
                                size_t strideY,
                                size_t padLeft,
                                size_t padRight,
                                size_t padTop,
                                size_t padBottom);

template void im2col_gpu<double>(double* stacked,
                                 double const* data,
                                 size_t width,
                                 size_t height,
                                 size_t depth,
                                 size_t windowWidth,
                                 size_t windowHeight,
                                 size_t strideX,
                                 size_t strideY,
                                 size_t padLeft,
                                 size_t padRight,
                                 size_t padTop,
                                 size_t padBottom);

/* ---------------------------------------------------------------- */
/*                                                     col2im (GPU) */
/* ---------------------------------------------------------------- */

template <typename T>
__global__ void col2im_gpu_kernel(T* data,
                                  T const* stacked,
                                  const int numPatchesX,
                                  const int numPatchesY,
                                  const int dataVolume,
                                  const int width,
                                  const int height,
                                  const int depth,
                                  const int windowWidth,
                                  const int windowHeight,
                                  const int strideX,
                                  const int strideY,
                                  const int padLeft,
                                  const int padTop)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < dataVolume)
  {
    T accumulator = 0 ;
    /*
     This kernel accumulates on data[index] all elements in stacked
     that receive copies of data[index] in im2col.
     
     Consider coordinate (x_data,y_data) in the input image. Relative to patch
     (x,y), this has offset
     
     u = x_data - (x * strideX - padLeft)
     v = y_data - (y * strideY - padRight)
     
     In particular, (x_data,y_data) is contained (and hence contributes)
     to patch (x,y) if, and only if,
     
     0 <= u < windowWidth  <==>  1) x_data >= x * strideX - padLeft
                                 2) x_data <  x * strideX - padLeft + windowWidth
     
     and similar for y.
     
     Hence, the patches that contribute to (x_data,y_data) are given
     by indexes (x,y) such that
     
     (x_data + padLeft - windowWidth)/stride < x
         <= (x_data + padLeft)/stride
     
     or, accounting for the boundaries,

       x1 <= x <= x2, such that
         x1 = max(0,  1 + floor(x_data + padLeft - windowWidth)/stride),
         x2 = min(numPatchesX-1,  floor(x_data + padLeft)/stride),
     
     and similar for y.
     
     Note that (x_data + padLeft - windowWidth) may be negative. In this case,
     the C convention for rounding division towards zero fails to compute
     the floor() properly. Instead, we check this case explicitly and set
     */

    int x_data = index ;
    int y_data = x_data / width ;
    int z = y_data / height ;
    x_data %= width ;
    y_data %= height ;

    int dx = x_data + padLeft - windowWidth ;
    int dy = y_data + padTop - windowHeight ;
    int x1 = (dx >= 0) ? dx/strideX + 1 : 0 ;
    int y1 = (dy >= 0) ? dy/strideY + 1 : 0 ;
    int x2 = min((x_data + padLeft) / strideX, numPatchesX - 1) ;
    int y2 = min((y_data + padTop) / strideY, numPatchesY - 1) ;

    /*
     Knowing which patches (x,y) contribute to (x_data,y_data) is not enough;
     we need to determine the specific element within each patch. This
     is given by the offset as given above:
     
     u(x) = x_data - (x * strideX - padLeft)
     v(y) = y_data - (y * strideY - padRight)
     
     Now we can compute the indices of the elements of stacked[] to accumulate:
     
     stackedIndex(x,y) =
         (y * numPatchesX + x) +                 // column offset
         ((z * windowHeight + v(y)) * windowWidth + u(x)) *  // within patch offset
            (numPatchesX*numPatchesY)

     Substituting the expression fo u(x), we find

     stackedIndex(x,y) =
         = (y * numPatchesX + x)
         + ((z * windowHeight + y_data + padTop) * windowWidth + x_data + padLeft)
           * (numPatchesX*numPatchesY)
         - ((y * strideY) * windowWidth + x * strideX)
           * (numPatchesX*numPatchesY)
         = (z * windowHeight + y_data + padTop) * windowWidth + x_data + padLeft)
         + x * (1 - strideX*numPatchesY*numPatchesX)
         + y * (1 - strideY*numPatchesY*windowWidth)*numPatchesX ;

     */

    int deltax = (1 - strideX * numPatchesY * numPatchesX) ;
    int deltay = (1 - strideY * numPatchesY * windowWidth) * numPatchesX ;
    stacked += ((z * windowHeight + y_data + padTop) * windowWidth + (x_data + padLeft)) * (numPatchesX*numPatchesY) ;

    for (int y = y1 ; y <= y2 ; ++ y) {
      for (int x = x1 ; x <= x2 ; ++ x) {
        accumulator += stacked[y * deltay + x * deltax];
      }
    }
    data[index] = accumulator;
  }
}

template <typename T>
void col2im_gpu(T* data,
                T const* stacked,
                size_t width,
                size_t height,
                size_t depth,
                size_t windowWidth,
                size_t windowHeight,
                size_t strideX,
                size_t strideY,
                size_t padLeft,
                size_t padRight,
                size_t padTop,
                size_t padBottom)
{
  /*
   each kernel integrates all contributions to a particular element
   of data.
   */
  int numPatchesX = (width + (padLeft + padRight) - windowWidth)/strideX + 1 ;
  int numPatchesY = (height + (padTop + padBottom) - windowHeight)/strideY + 1 ;
  int dataVolume = width * height * depth ;

  col2im_gpu_kernel<T>
  <<< divideUpwards(dataVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (data,
   stacked,
   numPatchesX,
   numPatchesY,
   dataVolume,
   width, height, depth,
   windowWidth, windowHeight,
   strideX, strideY,
   padLeft, padTop) ;

  if (hipPeekAtLastError() != hipSuccess) {
    std::cout
    <<"col2im: CUDA kernel error ("
    <<hipGetErrorString(hipPeekAtLastError())
    <<")"<<std::endl ;
  }
}

template void col2im_gpu<float>(float* data,
                                float const* stacked,
                                size_t width,
                                size_t height,
                                size_t depth,
                                size_t windowWidth,
                                size_t windowHeight,
                                size_t strideX,
                                size_t strideY,
                                size_t padLeft,
                                size_t padRight,
                                size_t padTop,
                                size_t padBottom);

template void col2im_gpu<double>(double* data,
                                 double const* stacked,
                                 size_t width,
                                 size_t height,
                                 size_t depth,
                                 size_t windowWidth,
                                 size_t windowHeight,
                                 size_t strideX,
                                 size_t strideY,
                                 size_t padLeft,
                                 size_t padRight,
                                 size_t padTop,
                                 size_t padBottom);
