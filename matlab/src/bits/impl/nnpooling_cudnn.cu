// @file nnpooling_blas.cu
// @brief Pooling block CuDNN-based implementation.
// @author Andrea Vedaldi

/*
 Copyright (C) 2015-16 Andrea Vedaldi.
 All rights reserved.

 This file is part of the VLFeat library and is made available under
 the terms of the BSD license (see the COPYING file).
 */

#if !defined(ENABLE_GPU) | !defined(ENABLE_CUDNN)
#error "nnpooling_cudnn.hpp cannot be compiled without GPU and CUDNN support."
#endif

#include "nnpooling_cudnn.hpp"
#include "cudnnhelper.hpp"
#include "../datacu.hpp"
#include <assert.h>

using namespace vl ;

#define CHECK(x) \
{ \
cudnnError = x ; \
if (cudnnError != HIPDNN_STATUS_SUCCESS) { \
error = context.setError(context.getCudaHelper().catchCudnnError(cudnnError, \
STRINGIZE(__LINE__) ":" STRINGIZE(__FILE__))) ; \
goto done ; \
} }

/* ---------------------------------------------------------------- */
/*                                         nnpooling_cudnn::forward */
/* ---------------------------------------------------------------- */


namespace vl { namespace impl {


  template<vl::Type dataType>
  vl::Error
  nnpooling_cudnn<dataType>::forward(Context& context,
                                     Tensor output,
                                     Tensor data,
                                     PoolingMethod method,
                                     int poolHeight, int poolWidth,
                                     int strideY, int strideX,
                                     int padTop, int padBottom,
                                     int padLeft, int padRight)
  {
    assert(output) ;
    assert(data) ;

    typedef typename DataTypeTraits<dataType>::type type ;

    hipdnnTensorDescriptor_t outputDesc, dataDesc ;
    hipdnnPoolingDescriptor_t poolingDesc ;
    bool outputDescInitialized = false ;
    bool dataDescInitialized = false ;
    bool poolingDescInitialized = false ;

    if (padLeft != padRight) return vl::vlErrorUnsupported ;
    if (padTop != padBottom) return vl::vlErrorUnsupported ;

    if (method == vlPoolingAverage && (padLeft > 0 | padRight > 0)) {
      /* This seems like a bug in CUDNN? */
      return vl::vlErrorUnsupported ;
    }

    hipdnnDataType_t cudnnDataType = DataTypeToCudnn<dataType>::id ;
    vl::Type dynDataType = output.getDataType() ;
    assert(dynDataType == dataType) ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::Error error = vl::vlSuccess ;
    hipdnnHandle_t handle ;

    // Get CuDNN
    CHECK(context.getCudaHelper().getCudnnHandle(&handle)) ;

    // Get tensor descripotrs
    CHECK(hipdnnCreateTensorDescriptor(&outputDesc)) ;
    outputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(outputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     output.getSize(), // sizes
                                     output.getDepth(),
                                     output.getWidth(),
                                     output.getHeight())) ;

    CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(dataDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     data.getSize(),
                                     data.getDepth(),
                                     data.getWidth(),
                                     data.getHeight())) ;

    CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc)) ;
    poolingDescInitialized = true ;
    CHECK(hipdnnSetPooling2dDescriptor(poolingDesc,
                                      (method == vl::vlPoolingAverage) ? HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING : HIPDNN_POOLING_MAX,
                                      IF_CUDNN_GE5(HIPDNN_NOT_PROPAGATE_NAN COMMA)
                                      poolWidth, poolHeight,
                                      padLeft, padTop,
                                      strideX, strideY)) ;

    // Perform convolution for each filter group
    {
      type alpha = 1.0f ;
      type beta = 0.0f ;
      CHECK(hipdnnPoolingForward(handle,
                                poolingDesc,
                                &alpha,
                                dataDesc, data.getMemory(),
                                &beta,
                                outputDesc, output.getMemory())) ;
    }

    /* cleanup */
  done:
    if (poolingDescInitialized) { hipdnnDestroyPoolingDescriptor(poolingDesc) ; }
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    if (outputDescInitialized) { hipdnnDestroyTensorDescriptor(outputDesc) ; }
    return context.passError(error, "nnpooling_cudnn::forward") ;
  }

  /* ---------------------------------------------------------------- */
  /*                                        nnpooling_cudnn::backward */
  /* ---------------------------------------------------------------- */

  template<vl::Type dataType>
  vl::Error
  nnpooling_cudnn<dataType>::backward(Context& context,
                                      Tensor derData,
                                      Tensor data,
                                      Tensor output,
                                      Tensor derOutput,
                                      vl::PoolingMethod method,
                                      int poolHeight, int poolWidth,
                                      int strideY, int strideX,
                                      int padTop, int padBottom,
                                      int padLeft, int padRight)
  {
    assert(derData) ;
    assert(data) ;
    assert(output) ;
    assert(derOutput) ;

    typedef typename DataTypeTraits<dataType>::type type ;

    hipdnnTensorDescriptor_t outputDesc, dataDesc ;
    hipdnnPoolingDescriptor_t poolingDesc ;
    bool outputDescInitialized = false ;
    bool dataDescInitialized = false ;
    bool poolingDescInitialized = false ;

    if (padLeft != padRight) return vl::vlErrorUnsupported ;
    if (padTop != padBottom) return vl::vlErrorUnsupported ;

    if (method == vlPoolingAverage && (padLeft > 0 | padRight > 0)) {
      /* This seems like a bug in CuDNN? */
      return vl::vlErrorUnsupported ;
    }

    hipdnnDataType_t cudnnDataType = DataTypeToCudnn<dataType>::id ;
    vl::Type dynDataType = output.getDataType() ;
    assert(dynDataType == dataType) ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::Error error = vl::vlSuccess ;
    hipdnnHandle_t handle ;

    // Get CuDNN
    CHECK(context.getCudaHelper().getCudnnHandle(&handle)) ;

    // Get tensor descripotrs
    CHECK(hipdnnCreateTensorDescriptor(&outputDesc)) ;
    outputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(outputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     output.getSize(), // sizes
                                     output.getDepth(),
                                     output.getWidth(),
                                     output.getHeight())) ;

    CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(dataDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     data.getSize(),
                                     data.getDepth(),
                                     data.getWidth(),
                                     data.getHeight())) ;

    CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc)) ;
    poolingDescInitialized = true ;
    CHECK(hipdnnSetPooling2dDescriptor(poolingDesc,
                                      (method == vl::vlPoolingAverage) ? HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING : HIPDNN_POOLING_MAX,
                                      IF_CUDNN_GE5(HIPDNN_NOT_PROPAGATE_NAN COMMA)
                                      poolWidth, poolHeight,
                                      padLeft, padTop,
                                      strideX, strideY)) ;

    // Perform convolution for each filter group
    {
      type alpha = 1.0f ;
      type beta = 0.0f ;
      CHECK(hipdnnPoolingBackward(handle,
                                 poolingDesc,
                                 &alpha,
                                 outputDesc, (type const*)output.getMemory(),
                                 outputDesc, (type const*)derOutput.getMemory(),
                                 dataDesc, (type const*)data.getMemory(),
                                 &beta,
                                 dataDesc, (type*)derData.getMemory())) ;
    }

    /* cleanup */
  done:
    if (poolingDescInitialized) { hipdnnDestroyPoolingDescriptor(poolingDesc) ; }
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    if (outputDescInitialized) { hipdnnDestroyTensorDescriptor(outputDesc) ; }
    return context.passError(error, __func__) ;
  }
  
} }

// Instantiations
template struct vl::impl::nnpooling_cudnn<vl::vlTypeFloat> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::nnpooling_cudnn<vl::vlTypeDouble> ;
#endif



