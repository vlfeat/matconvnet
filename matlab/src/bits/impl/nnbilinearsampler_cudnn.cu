// @file nnbilinearsampler_cudnn.cu
// @brief BilinearSampler CuDNN-based implementation.
// @author Ankush Gupta, Andrea Vedaldi

/*
Copyright (C) 2016- Ankush Gupta, Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#if !defined(ENABLE_GPU) || !defined(ENABLE_CUDNN)
#error "bilinearsampler_cudnn.cu can only be compiled with GPU and CUDNN (v5 or higher) support."
#endif

#include "nnbilinearsampler_cudnn.hpp"
#include "cudnnhelper.hpp"
#include "../datacu.hpp"
#include <assert.h>
#include <algorithm>

#if CUDNN_VERSION < 5000
#warning "bilinearsampler_cudnn.cu will be disabled as it requires CUDNN v5 or higher."

namespace vl { namespace impl {
  template<vl::Type dataType>
  vl::Error
  vl::impl::nnbilinearsampler_cudnn<dataType>::forward(Context& context,
                                                       Tensor output,
                                                       Tensor data,
                                                       Tensor grid)
  {
    return vl::vlErrorUnsupported ;
  }

  template<vl::Type dataType>
  vl::Error
  vl::impl::nnbilinearsampler_cudnn<dataType>::backward(Context& context,
                                                        Tensor derData,
                                                        Tensor derGrid,
                                                        Tensor data,
                                                        Tensor grid,
                                                        Tensor derOutput)
  {
    return vl::vlErrorUnsupported ;
  }
}}
#else

using namespace vl ;

// check if the descriptors, etc. were successfully created:
#define CHECK(x) \
{ \
cudnnError = x ; \
if (cudnnError != HIPDNN_STATUS_SUCCESS) { \
error = context.setError(context.getCudaHelper().catchCudnnError(cudnnError, \
STRINGIZE(__FILE__) ":" STRINGIZE(__LINE__))) ; \
goto done ; \
} }

/* ---------------------------------------------------------------- */
/*                                    bilinearsampler_forward_cudnn */
/* ---------------------------------------------------------------- */
namespace vl { namespace impl {

  template<vl::Type dataType>
  vl::Error
  vl::impl::nnbilinearsampler_cudnn<dataType>::forward(Context& context,
                                                       Tensor output,
                                                       Tensor data,
                                                       Tensor grid)
  {
    assert(output) ;
    assert(data) ;
    assert(grid) ;

    typedef typename DataTypeTraits<dataType>::type type ;

    hipdnnTensorDescriptor_t outputDesc, dataDesc ;
    cudnnSpatialTransformerDescriptor_t samplerDesc ;
    bool outputDescInitialized = false ;
    bool dataDescInitialized = false ;
    bool samplerDescInitialized = false ;

    // get the sizes:
    int inCardinality = data.getSize();
    int inDepth = data.getDepth();
    int inHeight = data.getHeight();
    int inWidth = data.getWidth();

    int outCardinality = output.getSize();
    int outDepth = output.getDepth();
    int outWidth = output.getWidth();
    int outHeight = output.getHeight();

    hipdnnDataType_t cudnnDataType = DataTypeToCudnn<dataType>::id ;
    vl::Type dynDataType = output.getDataType() ;
    assert(dynDataType == dataType) ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::Error error = vl::vlSuccess ;
    hipdnnHandle_t handle ;

    // get number of transforms/image == groupSize:
    int groupSize = outCardinality / inCardinality ;
    int dimOut[4] = { 1, outDepth, outWidth, outHeight } ; // one-image

    // Get CuDNN
    CHECK(context.getCudaHelper().getCudnnHandle(&handle)) ;

    // Get tensor descriptors:
    CHECK(hipdnnCreateTensorDescriptor(&outputDesc)) ;
    outputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptorEx(outputDesc,
                                       cudnnDataType,
                                       1, outDepth, outWidth, outHeight, // sizes: n,c,w,h
                                       outHeight * outWidth * outDepth, //strides
                                       outHeight * outWidth,
                                       outHeight,
                                       1)) ;

    CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptorEx(dataDesc,
                                       cudnnDataType,
                                       1, inDepth, inWidth, inHeight, // sizes: n,c,w,h
                                       inHeight * inWidth * inDepth, //strides
                                       inHeight * inWidth,
                                       inHeight,
                                       1)) ;

    // Get bilinear-sampler descriptor:
    CHECK(cudnnCreateSpatialTransformerDescriptor(&samplerDesc)) ;
    samplerDescInitialized = true ;
    CHECK(cudnnSetSpatialTransformerNdDescriptor(samplerDesc,
                                                 CUDNN_SAMPLER_BILINEAR,
                                                 cudnnDataType,
                                                 4,
                                                 dimOut)) ;

    {
      type alpha = 1.0f ;
      type beta = 0.0f ;
      const ptrdiff_t dataOffset = inHeight * inWidth * inDepth ;
      const ptrdiff_t gridOffset = 2 * outWidth * outHeight ;
      const ptrdiff_t outOffset = outHeight * outWidth * outDepth ;
      type const* data_ptr = (type const*) data.getMemory() ;
      type const* grid_ptr = (type const*) grid.getMemory() ;
      type * out_ptr = (type *) output.getMemory() ;

      for (int im=0; im < inCardinality; im++) {
        for (int ig=0; ig < groupSize; ig++) {
          cudnnSpatialTfSamplerForward(handle,
                                       samplerDesc,
                                       &alpha,
                                       dataDesc, data_ptr,
                                       grid_ptr,
                                       &beta,
                                       outputDesc, out_ptr) ;
          grid_ptr += gridOffset ;
          out_ptr += outOffset ;
        }
        data_ptr += dataOffset ;
      }
    }

  done:
    if (samplerDescInitialized) { cudnnDestroySpatialTransformerDescriptor(samplerDesc) ; }
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    if (outputDescInitialized) { hipdnnDestroyTensorDescriptor(outputDesc) ; }
    return context.passError(error, __func__) ;
  }

  /* ---------------------------------------------------------------- */
  /*                                   bilinearsampler_backward_cudnn */
  /* ---------------------------------------------------------------- */
  template<vl::Type dataType>
  vl::Error
  vl::impl::nnbilinearsampler_cudnn<dataType>::backward(Context& context,
                                                        Tensor derData,
                                                        Tensor derGrid,
                                                        Tensor data,
                                                        Tensor grid,
                                                        Tensor derOutput)
  {
    typedef typename DataTypeTraits<dataType>::type type ;

    /* no derDataDesc needed as same as dataDesc <-- nice! */
    hipdnnTensorDescriptor_t dataDesc, derOutputDesc ;
    cudnnSpatialTransformerDescriptor_t samplerDesc ;
    bool dataDescInitialized = false ;
    bool derOutputDescInitialized = false ;
    bool samplerDescInitialized = false ;

    // get the sizes:
    int inCardinality = data.getSize();
    int inDepth = data.getDepth();
    int inHeight = data.getHeight();
    int inWidth = data.getWidth();

    int outCardinality = derOutput.getSize();
    int outDepth = derOutput.getDepth();
    int outWidth = derOutput.getWidth();
    int outHeight = derOutput.getHeight();

    hipdnnDataType_t cudnnDataType = DataTypeToCudnn<dataType>::id ;
    vl::Type dynDataType = derOutput.getDataType() ;
    assert(dynDataType == dataType) ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::Error error = vl::vlSuccess ;
    hipdnnHandle_t handle ;

    // get number of transforms/image == groupSize:
    int groupSize = outCardinality / inCardinality;
    int dimOut[4] = { 1, outDepth, outWidth, outHeight };

    // Get CuDNN
    CHECK(context.getCudaHelper().getCudnnHandle(&handle)) ;


    // Get tensor descriptors:
    CHECK(hipdnnCreateTensorDescriptor(&derOutputDesc)) ;
    derOutputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptorEx(derOutputDesc,
                                       cudnnDataType,
                                       1, outDepth, outWidth, outHeight, // sizes: n,c,w,h
                                       outHeight * outWidth * outDepth, //strides
                                       outHeight * outWidth,
                                       outHeight,
                                       1)) ;

    CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptorEx(dataDesc,
                                       cudnnDataType,
                                       1, inDepth, inWidth, inHeight, // sizes: n,c,w,h
                                       inHeight * inWidth * inDepth, //strides
                                       inHeight * inWidth,
                                       inHeight,
                                       1)) ;

    // Get bilinear-sampler descriptor:
    CHECK(cudnnCreateSpatialTransformerDescriptor(&samplerDesc)) ;
    samplerDescInitialized = true ;
    CHECK(cudnnSetSpatialTransformerNdDescriptor(samplerDesc,
                                                 CUDNN_SAMPLER_BILINEAR,
                                                 cudnnDataType,
                                                 4,
                                                 dimOut));
    /* do the work */
    {
      type alpha = 1.0f ;
      type dataBeta = 1.0f ; // assuming that the derData has been initialized to zero
      type gridBeta = 0.0f ;
      const ptrdiff_t dataOffset = inHeight * inWidth * inDepth ;
      const ptrdiff_t gridOffset = 2 * outWidth * outHeight ;
      const ptrdiff_t outOffset = outHeight * outWidth * outDepth ;
      type const* data_ptr = (type const*) data.getMemory() ;
      type * derData_ptr = (type *) derData.getMemory() ;
      type const* grid_ptr = (type const*) grid.getMemory() ;
      type * derGrid_ptr = (type *) derGrid.getMemory() ;
      type * derOut_ptr = (type *) derOutput.getMemory() ;

      for (int im=0; im < inCardinality; im++) {
        for (int ig=0; ig < groupSize; ig++) {
        cudnnSpatialTfSamplerBackward(handle,
                                      samplerDesc,
                                      &alpha,
                                      dataDesc, data_ptr,
                                      &dataBeta,
                                      dataDesc, derData_ptr,
                                      &alpha,
                                      derOutputDesc, derOut_ptr,
                                      grid_ptr,
                                      &gridBeta,
                                      derGrid_ptr) ;
          grid_ptr += gridOffset ;
          derGrid_ptr += gridOffset ;
          derOut_ptr += outOffset ;
        }
        data_ptr += dataOffset ;
        derData_ptr += dataOffset ;
      }
    }

  /* cleanup */
  done:
    if (samplerDescInitialized) { cudnnDestroySpatialTransformerDescriptor(samplerDesc) ; }
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    if (derOutputDescInitialized) { hipdnnDestroyTensorDescriptor(derOutputDesc) ; }
    return context.passError(error, __func__) ;
  }
}}

#endif // CUDNN >= v5.0

// Instantiations
template struct vl::impl::nnbilinearsampler_cudnn<vl::vlTypeFloat> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::nnbilinearsampler_cudnn<vl::vlTypeDouble> ;
#endif
