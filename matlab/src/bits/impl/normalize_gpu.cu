#include "hip/hip_runtime.h"
// @file normalize_gpu.c
// @brief Normalize block implementation (GPU)
// @author Andrea Vedaldi

/*
Copyright (C) 2014-16 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "normalize.hpp"
#include "../datacu.hpp"
#include <assert.h>
#include <float.h>

/* ---------------------------------------------------------------- */
/*                                         normalize_forward_kernel */
/* ---------------------------------------------------------------- */

#undef xat
#undef yat
#undef zat
#define xat(t) x[(t) * offset]
#define yat(t) y[(t) * offset]
#define zat(t) z[(t) * offset]

#define __powf powf

template<typename T> __global__ void
normalize_forward_kernel
(T* output,
 T const* data,
 int width,
 int height,
 int depth,
 int num,
 int normDepth,
 T kappa, T alpha, T beta)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < width*height*num) {
    int u0 = index ;
    int v0 = u0 / width ;
    int k0 = v0 / height ;
    u0 %= width ;
    v0 %= height ;

    int m1 = ((signed)normDepth-1)/2 ;
    int m2 = normDepth - m1 - 1 ;
    int offset = width*height ;
    int t ;
    T const* x = data + u0 + (v0 + k0 * (depth*height)) * width ;
    T* y = output + u0 + (v0 + k0 * (depth*height)) * width ;
    T acc = 0 ;
    for (t = -m2 ; t < (signed)depth ; ++t) {
      T ap = 0 ;
      T am = 0 ;
      if (t-m1-1 >= 0) { am = xat(t-m1-1) ; }
      if (t+m2 < depth) { ap = xat(t+m2) ; }
      acc += ap*ap - am*am ;
      if (0 <= t && t < depth) {
        yat(t) = xat(t) * __powf(kappa + alpha * acc, -beta) ;
      }
    }
  }
}

/* ---------------------------------------------------------------- */
/*                                        normalize_backward_kernel */
/* ---------------------------------------------------------------- */

template<typename T> __global__ void
normalize_backward_kernel
(T* output,
 T const* data,
 T const* dzdy,
 int width,
 int height,
 int depth,
 int num,
 int normDepth,
 T kappa, T alpha, T beta)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < width*height*num) {
    int u0 = index ;
    int v0 = u0 / width ;
    int k0 = v0 / height ;
    u0 %= width ;
    v0 %= height ;

    int m1 = ((signed)normDepth-1)/2 ;
    int m2 = normDepth - m1 - 1 ;
    int offset = width*height ;
    T ab2 = 2*alpha*beta ;
    int t, q ;
    T const* x = data + u0 + (v0 + k0 * (depth*height)) * width ;
    T* y = output + u0 + (v0 + k0 * (depth*height)) * width ;
    T const* z = dzdy + u0 + (v0 + k0 * (depth*height)) * width ;
    T acc = 0 ;
    for (t = 0 ; t < (signed)depth ; ++t) {
      yat(t) = 0 ;
    }
    for (t = -m2 ; t < (signed)depth ; ++t) {
      int q1 = t-m1 ;
      int q2 = t+m2 ;
      T ap = 0 ;
      T am = 0 ;
      if (t-m1-1 >= 0) { am = xat(t-m1-1) ; } else { q1 = 0 ; }
      if (t+m2 < depth) { ap = xat(t+m2) ; } else { q2 = depth - 1 ; }
      acc += ap*ap - am*am ;
      T L = kappa + alpha * acc ;
      T Lbeta = __powf(L, -beta) ;
      T Lbeta1 = Lbeta / L ;

      if (0 <= t && t < depth) {
        yat(t) += zat(t) * Lbeta ;
        for (q = q1 ; q <= q2 ; ++ q) {
          yat(q) -= zat(t) * xat(t) * xat(q) * ab2 * Lbeta1 ;
        }
      }
    }
  }
}

/* ---------------------------------------------------------------- */
/*                                                          drivers */
/* ---------------------------------------------------------------- */

namespace vl { namespace impl {


  template<typename type>
  struct lrn<vl::GPU, type>
  {

    /* ------------------------------------------------------------ */
    /*                                                      forward */
    /* ------------------------------------------------------------ */

    static vl::Error
    forward(type * output,
            type  const* data,
            size_t width,
            size_t height,
            size_t depth,
            size_t size,
            size_t normDepth,
            type kappa, type alpha, type beta)
    {
      normalize_forward_kernel<type >
      <<< divideUpwards(width*height*size, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
      (output, data, width, height, depth, size, normDepth, kappa, alpha, beta) ;

      hipError_t status = hipPeekAtLastError() ;
      return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
    }


    /* ------------------------------------------------------------ */
    /*                                                      forward */
    /* ------------------------------------------------------------ */

    static vl::Error
    backward(type * derData,
             type  const* data,
             type  const* derOutput,
             size_t width,
             size_t height,
             size_t depth,
             size_t size,
             size_t normDepth,
             type kappa, type alpha, type beta)
    {
      normalize_backward_kernel<type >
      <<< divideUpwards(width*height*size, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
      (derData, data, derOutput, width, height, depth, size, normDepth, kappa, alpha, beta) ;

      hipError_t status = hipPeekAtLastError() ;
      return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
    }

  } ;

} }

// Instantiations
template struct vl::impl::lrn<vl::GPU, float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::lrn<vl::GPU, double> ;
#endif



