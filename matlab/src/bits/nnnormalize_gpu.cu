#include "hip/hip_runtime.h"
// @file nnnormalize_gpu.c
// @brief Normalize block implementation (GPU)
// @author Andrea Vedaldi

/*
Copyright (C) 2014-17 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnnormalize.hpp"
#include "datacu.hpp"

// -------------------------------------------------------------------
//                                                             Helpers
// -------------------------------------------------------------------

#undef xat
#undef yat
#undef zat
#define xat(t) x[(t) * offset]
#define yat(t) y[(t) * offset]
#define zat(t) z[(t) * offset]

template<typename T> __global__ void
normalize_forward_kernel
(T* output,
 T const* data,
 int width,
 int height,
 int depth,
 int num,
 int normDepth,
 T kappa, T alpha, T beta)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < width*height*num) {
    int u0 = index ;
    int v0 = u0 / width ;
    int k0 = v0 / height ;
    u0 %= width ;
    v0 %= height ;

    int m1 = ((signed)normDepth-1)/2 ;
    int m2 = normDepth - m1 - 1 ;
    int offset = width*height ;
    int t ;
    T const* x = data + u0 + (v0 + k0 * (depth*height)) * width ;
    T* y = output + u0 + (v0 + k0 * (depth*height)) * width ;
    T acc = 0 ;
    for (t = -m2 ; t < (signed)depth ; ++t) {
      T ap = 0 ;
      T am = 0 ;
      if (t-m1-1 >= 0) { am = xat(t-m1-1) ; }
      if (t+m2 < depth) { ap = xat(t+m2) ; }
      acc += ap*ap - am*am ;
      if (0 <= t && t < depth) {
        yat(t) = xat(t) * powf(kappa + alpha * acc, -beta) ;
      }
    }
  }
}

template<typename T> __global__ void
normalize_backward_kernel
(T* output,
 T const* data,
 T const* dzdy,
 int width,
 int height,
 int depth,
 int num,
 int normDepth,
 T kappa, T alpha, T beta)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < width*height*num) {
    int u0 = index ;
    int v0 = u0 / width ;
    int k0 = v0 / height ;
    u0 %= width ;
    v0 %= height ;

    int m1 = ((signed)normDepth-1)/2 ;
    int m2 = normDepth - m1 - 1 ;
    int offset = width*height ;
    T ab2 = 2*alpha*beta ;
    int t, q ;
    T const* x = data + u0 + (v0 + k0 * (depth*height)) * width ;
    T* y = output + u0 + (v0 + k0 * (depth*height)) * width ;
    T const* z = dzdy + u0 + (v0 + k0 * (depth*height)) * width ;
    T acc = 0 ;
    for (t = 0 ; t < (signed)depth ; ++t) {
      yat(t) = 0 ;
    }
    for (t = -m2 ; t < (signed)depth ; ++t) {
      int q1 = t-m1 ;
      int q2 = t+m2 ;
      T ap = 0 ;
      T am = 0 ;
      if (t-m1-1 >= 0) { am = xat(t-m1-1) ; } else { q1 = 0 ; }
      if (t+m2 < depth) { ap = xat(t+m2) ; } else { q2 = depth - 1 ; }
      acc += ap*ap - am*am ;
      T L = kappa + alpha * acc ;
      T Lbeta = powf(L, -beta) ;
      T Lbeta1 = Lbeta / L ;

      if (0 <= t && t < depth) {
        yat(t) += zat(t) * Lbeta ;
        for (q = q1 ; q <= q2 ; ++ q) {
          yat(q) -= zat(t) * xat(t) * xat(q) * ab2 * Lbeta1 ;
        }
      }
    }
  }
}

// -------------------------------------------------------------------
//                                                         Forward GPU
// -------------------------------------------------------------------

template<vl::DataType dataType>
struct LRNForward<vl::VLDT_GPU, dataType>
{
  vl::ErrorCode operator()(vl::nn::LRN &op,
                           vl::Tensor &output,
                           vl::Tensor const &input)
  {
    typedef typename vl::DataTypeTraits<dataType>::type type ;
    auto width = output.getWidth() ;
    auto height = output.getHeight() ;
    auto depth = output.getDepth() ;
    auto size = output.getSize() ;
    auto inputData = (type const*)input.getMemory() ;
    auto outputData = (type*)output.getMemory() ;

    normalize_forward_kernel<type>
    <<< divideAndRoundUp(width*height*size, (size_t)VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
    (outputData, inputData, width, height, depth, size,
     op.normDepth, op.kappa, op.alpha, op.beta) ;

    hipError_t status = hipPeekAtLastError() ;
    return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
  }
} ;

// -------------------------------------------------------------------
//                                                        Backward GPU
// -------------------------------------------------------------------

template<vl::DataType dataType>
struct LRNBackward<vl::VLDT_GPU, dataType>
{
  vl::ErrorCode operator()(vl::nn::LRN &op,
                           vl::Tensor &derInput,
                           vl::Tensor const &input,
                           vl::Tensor const &derOutput)
  {
    typedef typename vl::DataTypeTraits<dataType>::type type ;
    auto width = derOutput.getWidth() ;
    auto height = derOutput.getHeight() ;
    auto depth = derOutput.getDepth() ;
    auto size = derOutput.getSize() ;
    auto inputData = (type const*)input.getMemory() ;
    auto derOutputData = (type const*)derOutput.getMemory() ;
    auto derInputData = (type*)derInput.getMemory() ;

    normalize_backward_kernel<type >
    <<< divideAndRoundUp(width*height*size, (size_t)VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
    (derInputData, inputData, derOutputData, width, height, depth, size,
     op.normDepth, op.kappa, op.alpha, op.beta) ;

    hipError_t status = hipPeekAtLastError() ;
    return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
  }
} ;



