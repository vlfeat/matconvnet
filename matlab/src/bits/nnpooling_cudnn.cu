// @file nnpooling_cudnn.cu
// @brief Pooling layer CuDNN.
// @author Andrea Vedaldi

/*
Copyright (C) 2015-17 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnpooling.hpp"
#include "datacu.hpp"
#include "impl/cudnnhelper.hpp"
#include <cassert>

using namespace std ;
using namespace vl ;
using namespace vl::nn ;
using namespace vl::impl ;

#define CHECK(x) \
{ \
cudnnError = x ; \
if (cudnnError != HIPDNN_STATUS_SUCCESS) { \
error = op.context.setError(op.context.getCudaHelper().catchCudnnError(cudnnError, \
STRINGIZE(__LINE__) ":" STRINGIZE(__FILE__))) ; \
goto done ; \
} }

// -------------------------------------------------------------------
//                                                             Forward
// -------------------------------------------------------------------

template<DataType dataType>
struct PoolingForwardCudnn
{
  vl::ErrorCode operator()(Pooling &op,
                           Tensor &output,
                           Tensor const &input)
  {
    assert(output) ;
    assert(input) ;

    typedef typename DataTypeTraits<dataType>::type type ;

    hipdnnTensorDescriptor_t outputDesc, inputDesc ;
    hipdnnPoolingDescriptor_t poolingDesc ;
    bool outputDescInitialized = false ;
    bool inputDescInitialized = false ;
    bool poolingDescInitialized = false ;

    if (op.padLeft != op.padRight) return vl::VLE_Unsupported ;
    if (op.padTop != op.padBottom) return vl::VLE_Unsupported ;

    if (op.method == Pooling::Average && (op.padLeft > 0 | op.padRight > 0)) {
      // CuDNN bug? Skip.
      return vl::VLE_Unsupported ;
    }

    hipdnnDataType_t cudnnDataType = DataTypeToCudnn<dataType>::dataType ;
    vl::DataType dynDataType = output.getDataType() ;
    assert(dynDataType == dataType) ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN.
    CHECK(op.context.getCudaHelper().getCudnnHandle(&handle)) ;

    // Get tensor descriptors.
    CHECK(hipdnnCreateTensorDescriptor(&outputDesc)) ;
    outputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(outputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     output.getSize(),
                                     output.getDepth(),
                                     output.getWidth(),
                                     output.getHeight())) ;

    CHECK(hipdnnCreateTensorDescriptor(&inputDesc)) ;
    inputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(inputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     input.getSize(),
                                     input.getDepth(),
                                     input.getWidth(),
                                     input.getHeight())) ;

    CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc)) ;
    poolingDescInitialized = true ;
    CHECK(hipdnnSetPooling2dDescriptor(poolingDesc,
                                      (op.method == Pooling::Average) ? HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING : HIPDNN_POOLING_MAX,
                                      IF_CUDNN_GE5(HIPDNN_NOT_PROPAGATE_NAN COMMA)
                                      op.poolWidth, op.poolHeight,
                                      op.padLeft, op.padTop,
                                      op.strideX, op.strideY)) ;

    // Apply operator.
    {
      type alpha = 1.0f ;
      type beta = 0.0f ;
      CHECK(hipdnnPoolingForward(handle,
                                poolingDesc,
                                &alpha,
                                inputDesc, input.getMemory(),
                                &beta,
                                outputDesc, output.getMemory())) ;
    }

    // Finish.
  done:
    if (poolingDescInitialized) { hipdnnDestroyPoolingDescriptor(poolingDesc) ; }
    if (inputDescInitialized) { hipdnnDestroyTensorDescriptor(inputDesc) ; }
    if (outputDescInitialized) { hipdnnDestroyTensorDescriptor(outputDesc) ; }
    return op.context.passError(error, "nnpooling_cudnn::forward") ;
  }
} ;

// -------------------------------------------------------------------
//                                                            Backward
// -------------------------------------------------------------------

template<DataType dataType>
struct PoolingBackwardCudnn
{
  vl::ErrorCode operator()(Pooling &op,
                           Tensor &derInput,
                           Tensor const &input,
                           Tensor const &derOutput)
  {
    assert(derInput) ;
    assert(input) ;
    assert(derOutput) ;

    vl::ErrorCode error = VLE_Success ;
    typedef typename DataTypeTraits<dataType>::type type ;

    hipdnnTensorDescriptor_t derOutputDesc, inputDesc ;
    hipdnnPoolingDescriptor_t poolingDesc ;
    bool derOutputDescInitialized = false ;
    bool inputDescInitialized = false ;
    bool poolingDescInitialized = false ;

    if (op.padLeft != op.padRight) return vl::VLE_Unsupported ;
    if (op.padTop != op.padBottom) return vl::VLE_Unsupported ;

    if (op.method == Pooling::Average && (op.padLeft > 0 | op.padRight > 0)) {
      // CuDNN bug? Skip.
      return vl::VLE_Unsupported ;
    }

    hipdnnDataType_t cudnnDataType = DataTypeToCudnn<dataType>::dataType ;
    vl::DataType dynDataType = derInput.getDataType() ;
    assert(dynDataType == dataType) ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    hipdnnHandle_t handle ;
    Tensor output ;

    // CuDNN requires the output of the layer, so we recompute it here.
    size_t outputDataSize = derOutput.getNumElements() * sizeof(type) ;
    type * outputData = (type*)op.context.getWorkspace
    (vl::VLDT_GPU, outputDataSize) ;
    if (outputData == NULL) {
      error = VLE_OutOfMemory ;
      goto done ;
    }
    output = Tensor(derOutput, dataType, VLDT_GPU, outputData, outputDataSize) ;
    error = PoolingForwardCudnn<dataType>()(op,output,input) ;
    if (error != VLE_Success) {
      goto done ;
    }

    // Get CuDNN.
    CHECK(op.context.getCudaHelper().getCudnnHandle(&handle)) ;

    // Get tensor descripotrs.
    CHECK(hipdnnCreateTensorDescriptor(&derOutputDesc)) ;
    derOutputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(derOutputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     derOutput.getSize(),
                                     derOutput.getDepth(),
                                     derOutput.getWidth(),
                                     derOutput.getHeight())) ;

    CHECK(hipdnnCreateTensorDescriptor(&inputDesc)) ;
    inputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(inputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     input.getSize(),
                                     input.getDepth(),
                                     input.getWidth(),
                                     input.getHeight())) ;

    CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc)) ;
    poolingDescInitialized = true ;
    CHECK(hipdnnSetPooling2dDescriptor(poolingDesc,
                                      (op.method == Pooling::Average) ? HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING : HIPDNN_POOLING_MAX,
                                      IF_CUDNN_GE5(HIPDNN_NOT_PROPAGATE_NAN COMMA)
                                      op.poolWidth, op.poolHeight,
                                      op.padLeft, op.padTop,
                                      op.strideX, op.strideY)) ;

    // Apply operator.
    {
      type alpha = 1.0f ;
      type beta = 0.0f ;
      CHECK(hipdnnPoolingBackward(handle,
                                 poolingDesc,
                                 &alpha,
                                 derOutputDesc, outputData,
                                 derOutputDesc, (type const*)derOutput.getMemory(),
                                 inputDesc, (type const*)input.getMemory(),
                                 &beta,
                                 inputDesc, (type*)derInput.getMemory())) ;
    }

    // Finish.
  done:
    if (poolingDescInitialized) { hipdnnDestroyPoolingDescriptor(poolingDesc) ; }
    if (inputDescInitialized) { hipdnnDestroyTensorDescriptor(inputDesc) ; }
    if (derOutputDescInitialized) { hipdnnDestroyTensorDescriptor(derOutputDesc) ; }
    return op.context.passError(error, __func__) ;
  }
} ;



