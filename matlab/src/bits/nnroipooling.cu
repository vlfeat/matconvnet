#include "hip/hip_runtime.h"
// @file nnroipooling.cu
// @brief ROI pooling block
// @author Hakan Bilen
// @author Abishek Dutta
// @author Andrea Vedaldi

/*
Copyright (C) 2016-17 Hakan Bilen, Abishek Dutta, and Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnroipooling.hpp"
#include "impl/dispatcher.hpp"
#include <limits>
#include <cassert>
#include <cstring>
#include <cmath>
#include <iostream>

using namespace vl ;
using namespace vl::nn ;
using namespace vl::impl ;

template<DeviceType deviceType, DataType dataType> struct ROIPoolingForward ;
template<DeviceType deviceType, DataType dataType> struct ROIPoolingBackward ;

// -------------------------------------------------------------------
//                                                             Helpers
// -------------------------------------------------------------------

template <typename type>
struct acc_max
{
  inline acc_max(int poolHeight, int poolWidth, type derOutput = 0)
  :
  value(-std::numeric_limits<type>::infinity()),
  derOutput(derOutput),
  derDataActivePt(NULL)
  { }

  inline void accumulate_forward(type x) {
    value = std::max(value, x) ;
  }

  inline void accumulate_backward(type const* data, type* derDataPt) {
    type x = *data ;
    if (x > value) {
      value = x ;
      derDataActivePt = derDataPt ;
    }
  }

  inline type done_forward() const {
    return value ;
  }

  inline void done_backward() const {
    if (derDataActivePt) { *derDataActivePt += derOutput ; }
  }

  type value ;
  type derOutput ;
  type* derDataActivePt ;
} ;

template <typename type>
struct acc_sum
{
  inline acc_sum(int poolHeight, int poolWidth, type derOutput = 0)
  :
  value(0),
  scale(type(1)/type(poolHeight*poolWidth)),
  derOutput(derOutput)
  { }

  inline void accumulate_forward(type x) {
    value += x ;
  }

  inline void accumulate_backward(type const* data, type* derDataPt) {
    *derDataPt += derOutput * scale ;
  }

  inline type done_forward() const {
    return value * scale ;
  }

  inline void done_backward() const { }

  type value ;
  type derOutput ;
  type scale;
} ;

// -------------------------------------------------------------------
//                                                             Forward
// -------------------------------------------------------------------

template<DataType dataType, class Accumulator>
struct ROIPoolingForwardCPU
{
  vl::ErrorCode operator()(ROIPooling &op,
                           Tensor &pooled,
                           Tensor const &input,
                           Tensor const &rois)
  {
    typedef typename vl::DataTypeTraits<dataType>::type type ;
    auto numROIs = rois.getNumElements() / 5 ;
    auto height = input.getHeight() ;
    auto width = input.getWidth() ;
    auto depth = input.getDepth() ;
    auto size = input.getSize() ;
    auto roisData = (type const*)rois.getMemory() ;
    auto inputData = (type const*)input.getMemory() ;
    auto pooledData = (type*)pooled.getMemory() ;

    // For each ROI R = [t x1 y1 x2 y2].
    for (int roi = 0; roi < numROIs; ++roi) {

      // Apply scale and offset to each ROI coordinate.
      type u1_ = roisData[5 * roi + 1] ;
      type v1_ = roisData[5 * roi + 2] ;
      type u2_ = roisData[5 * roi + 3] ;
      type v2_ = roisData[5 * roi + 4] ;

      type u1 = op.transform[0] * u1_ + op.transform[2] * v1_ + op.transform[4] ;
      type v1 = op.transform[1] * u1_ + op.transform[3] * v1_ + op.transform[5] ;
      type u2 = op.transform[0] * u2_ + op.transform[2] * v2_ + op.transform[4] ;
      type v2 = op.transform[1] * u2_ + op.transform[3] * v2_ + op.transform[5] ;

      // First and last pixel of each ROI (rounded
      // for compatibility with the Caffe definition).
      int roi_image   = (int)roisData[5 * roi + 0];
      int roi_start_h = (int)::round(v1) - 1 ;
      int roi_start_w = (int)::round(u1) - 1 ;
      int roi_end_h   = (int)::round(v2) - 1 ;
      int roi_end_w   = (int)::round(u2) - 1 ;
      int roi_height  = std::max(roi_end_h - roi_start_h + 1, 1) ;
      int roi_width   = std::max(roi_end_w - roi_start_w + 1, 1) ;

      roi_image = std::min(std::max(roi_image - 1,0), (int)size - 1) ;
      type const * data_offset = inputData + (roi_image * depth) * (width*height) ;

      type bin_size_h = (double)roi_height / op.subdivisions[0] ;
      type bin_size_w = (double)roi_width / op.subdivisions[1] ;

      // For each feature channel.
      for (int z = 0; z < depth; ++z) {

        // For each column of tiles.
        for (int pw = 0; pw < op.subdivisions[1]; ++pw) {
          int wstart = (int)floor(((type)pw) * bin_size_w) ;
          int wend = (int)ceil(((type)(pw + 1)) * bin_size_w) ;
          wstart = std::min(std::max(wstart + roi_start_w, 0), (int)width) ;
          wend = std::min(std::max(wend + roi_start_w, 0), (int)width) ;

          // For each tile in a column.
          for (int ph = 0; ph < op.subdivisions[0]; ++ph) {
            int hstart = (int)floor(((type)ph) * bin_size_h) ;
            int hend = (int)ceil(((type)(ph + 1)) * bin_size_h) ;
            hstart = std::min(std::max(hstart + roi_start_h, 0), (int)height) ;
            hend = std::min(std::max(hend + roi_start_h, 0), (int)height) ;

            bool is_empty = (hend <= hstart) || (wend <= wstart);

            if (is_empty) {
              *pooledData++ = 0 ;
            }
            else {
              Accumulator acc(hend - hstart, wend - wstart) ;
              for (int w = wstart ; w < wend; ++w) {
                for (int h = hstart ; h < hend; ++h) {
                  const int index = w * height + h ;
                  acc.accumulate_forward(data_offset[index]) ;
                }
              }
              *pooledData++ = acc.done_forward() ;
            }
          } // end of ph
        } // end of pw
        data_offset += width*height;
      } // end of z
    } // end of n
    return VLE_Success ;
  }
} ;

template<DataType dataType>
struct ROIPoolingForward<VLDT_CPU,dataType>
{
  vl::ErrorCode operator()(ROIPooling &op,
                           Tensor &pooled,
                           Tensor const &input,
                           Tensor const &rois)
  {
    switch (op.method) {
      case ROIPooling::Max:
        return
        ROIPoolingForwardCPU<dataType,acc_max<typename vl::DataTypeTraits<dataType>::type> >
        ()(op,pooled,input,rois) ;
      case ROIPooling::Average:
        return
        ROIPoolingForwardCPU<dataType,acc_sum<typename vl::DataTypeTraits<dataType>::type> >
        ()(op,pooled,input,rois) ;
      default: return VLE_IllegalArgument ;
    }
  }
} ;

// -------------------------------------------------------------------
//                                                            Backward
// -------------------------------------------------------------------

template<DataType dataType, class Accumulator>
struct ROIPoolingBackwardCPU
{
  vl::ErrorCode operator()(ROIPooling &op,
                           Tensor &derInput,
                           Tensor const &input,
                           Tensor const &rois,
                           Tensor const &derOutput)
  {
    typedef typename vl::DataTypeTraits<dataType>::type type ;
    auto numROIs = rois.getNumElements() / 5 ;
    auto height = input.getHeight() ;
    auto width = input.getWidth() ;
    auto depth = input.getDepth() ;
    auto size = input.getSize() ;
    auto derInputData = (type*)derInput.getMemory() ;
    auto roisData = (type const*)rois.getMemory() ;
    auto inputData = (type const*)input.getMemory() ;
    auto derOutputData = (type const*)derOutput.getMemory() ;

    memset(derInputData, 0, derInput.getNumElements() * sizeof(type)) ;

    // For each ROI R = [t x1 y1 x2 y2].
    for (size_t roi = 0; roi < numROIs ; ++roi) {

      // Apply sacle and offset to each ROI coordinate.
      type u1_ = roisData[5 * roi + 1] ;
      type v1_ = roisData[5 * roi + 2] ;
      type u2_ = roisData[5 * roi + 3] ;
      type v2_ = roisData[5 * roi + 4] ;

      type u1 = op.transform[0] * u1_ + op.transform[2] * v1_ + op.transform[4] ;
      type v1 = op.transform[1] * u1_ + op.transform[3] * v1_ + op.transform[5] ;
      type u2 = op.transform[0] * u2_ + op.transform[2] * v2_ + op.transform[4] ;
      type v2 = op.transform[1] * u2_ + op.transform[3] * v2_ + op.transform[5] ;

      // First and last pixel of each ROI (rounded
      // for compatibility with the Caffe definition).
      int roi_image   = (int)roisData[5 * roi + 0];
      int roi_start_h = (int)::round(v1) - 1 ;
      int roi_start_w = (int)::round(u1) - 1 ;
      int roi_end_h   = (int)::round(v2) - 1 ;
      int roi_end_w   = (int)::round(u2) - 1 ;
      int roi_height = std::max(roi_end_h - roi_start_h + 1, 1) ;
      int roi_width = std::max(roi_end_w - roi_start_w + 1, 1) ;

      roi_image = std::min(std::max(roi_image - 1,0), (int)size - 1) ;
      type const * data_offset = inputData + (roi_image * depth) * (width*height);
      type * derInputData_offset = derInputData + (roi_image * depth) * (width*height);

      const type bin_size_h = (double)roi_height / op.subdivisions[0] ;
      const type bin_size_w = (double)roi_width / op.subdivisions[1] ;

      // For each feature channel.
      for (int z = 0; z < depth; ++z) {

        // For each column of tiles.
        for (int pw = 0; pw < op.subdivisions[1]; ++pw) {
          int wstart = (int)floor(((type)pw) * bin_size_w) ;
          int wend = (int)ceil(((type)(pw + 1)) * bin_size_w) ;
          wstart = std::min(std::max(wstart + roi_start_w, 0), (int)width) ;
          wend = std::min(std::max(wend + roi_start_w, 0), (int)width) ;

          // For each tile in a column.
          for (int ph = 0; ph < op.subdivisions[0]; ++ph) {
            int hstart = (int)floor(((type)ph) * bin_size_h) ;
            int hend = (int)ceil(((type)(ph + 1)) * bin_size_h) ;
            hstart = std::min(std::max(hstart + roi_start_h, 0), (int)height) ;
            hend = std::min(std::max(hend + roi_start_h, 0), (int)height) ;

            Accumulator acc(hend - hstart, wend - wstart, *derOutputData++) ;
            for (int w = wstart; w < wend; ++w) {
              for (int h = hstart; h < hend; ++h) {
                const int index = w * height + h ;
                acc.accumulate_backward(&data_offset[index],
                                        &derInputData_offset[index]) ;
              }
            }
            acc.done_backward() ;
          } // end of pw
        } // end of ph
        data_offset += width*height ;
        derInputData_offset += width*height ;
      } // end of z
    } // end of n

    return VLE_Success ;
  }
} ;

template<DataType dataType>
struct ROIPoolingBackward<VLDT_CPU,dataType>
{
  vl::ErrorCode operator()(ROIPooling &op,
                           Tensor &derInput,
                           Tensor const &input,
                           Tensor const &rois,
                           Tensor const &derOutput)
  {
    switch (op.method) {
      case ROIPooling::Max: return
        ROIPoolingBackwardCPU<dataType,acc_max<typename vl::DataTypeTraits<dataType>::type> >
        ()(op,derInput,input,rois,derOutput) ;
      case ROIPooling::Average: return
        ROIPoolingBackwardCPU<dataType,acc_sum<typename vl::DataTypeTraits<dataType>::type> >
        ()(op,derInput,input,rois,derOutput) ;
      default: return VLE_IllegalArgument ;
    }
  }
} ;

// -------------------------------------------------------------------
//                                                              Driver
// -------------------------------------------------------------------

#if ENABLE_GPU
#include "nnroipooling_gpu.cu"
#endif

ROIPooling::ROIPooling(Context &context,
                       std::array<int,2> subdivisions,
                       std::array<double,6> transform,
                       Method method) :
context(context),
subdivisions(subdivisions),
transform(transform),
method(method)
{ }

vl::ErrorCode
ROIPooling::forward(Tensor &output,
                    Tensor const &input,
                    Tensor const &rois)
{
  return dispatch<ROIPoolingForward>()(*this,output,input,rois) ;
}

vl::ErrorCode
ROIPooling::backward(Tensor &derInput,
                     Tensor const &input,
                     Tensor const &rois,
                     Tensor const &derOutput)
{
  return dispatch<ROIPoolingBackward>()(*this,derInput,input,rois,derOutput) ;
}
