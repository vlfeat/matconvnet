#include "hip/hip_runtime.h"
// @file nnpooling_gpu.cu
// @brief Pooling block GPU.
// @author Andrea Vedaldi
// @author Karel Lenc

/*
Copyright (C) 2014-17 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnpooling.hpp"
#include "datacu.hpp"
#include <assert.h>
#include <float.h>
#include <sm_20_atomic_functions.h>

// -------------------------------------------------------------------
//                                                 Max pooling helpers
// -------------------------------------------------------------------

template<typename T> __global__ void
pooling_max_kernel
(T* output,
 const T* data,
 const int outputWidth,
 const int outputHeight,
 const int outputVolume,
 const int width,
 const int height,
 const int poolWidth,
 const int poolHeight,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  int outputIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (outputIndex < outputVolume) {
    int px = outputIndex ;
    int py = px / outputWidth ;
    int pz = py / outputHeight ;
    px %= outputWidth ;
    py %= outputHeight ;
    data += pz * (width*height) ;

    int x1 = px * strideX - padLeft ;
    int y1 = py * strideY - padTop ;
    int x2 = min(x1 + poolWidth, width) ;
    int y2 = min(y1 + poolHeight, height) ;
    x1 = max(x1, 0) ;
    y1 = max(y1, 0) ;

    T bestValue = data[y1 * width + x1] ;
    for (int y = y1 ; y < y2 ; ++y) {
      for (int x = x1 ; x < x2 ; ++x) {
        bestValue = max(bestValue, data[y * width + x]) ;
      }
    }
    output[outputIndex] = bestValue ;
  }
}

#ifdef VLNN_CAFFELIKE_BPPOOL
// In order to be able to use this, BP would need to have access to both
// bottom data and output data (currently only passed bottom data...)
template <typename T> __global__ void
pooling_max_backward_with_output_data
(T* derData,
 const T* data,
 const T* output,
 const T* derOutput,
 const int nthreads,
 const int outputWidth,
 const int outputHeight,
 const int width,
 const int height,
 const int depth,
 const int poolWidth,
 const int poolHeight,
 const int strideX,
 const int strideY)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    // find out the local index
    // find out the local offset
    int x = index % width;
    int y = (index / width) % height;
    int z = (index / width / height) % depth;
    int py1 = (y < poolHeight) ? 0 : (y - poolHeight) / strideY + 1;
    int py2 = min(y / strideY + 1, outputHeight);
    int px1 = (x < poolWidth) ? 0 : (x - poolWidth) / strideX + 1;
    int px2 = min(x / strideX + 1, outputWidth);
    T gradient = 0;
    T datum = data[(z * height + y) * width + x];
    output += z * outputHeight * outputWidth;
    dzdy += z * outputHeight * outputWidth;
    for (int py = py1; py < py2; ++py) {
      for (int px = px1; px < px2; ++px) {
        gradient += dzdy[py * outputWidth + px] *
        (datum == output[py * outputWidth + px]);
      }
    }
    dzdx[index] = gradient;
  }
}
#endif

template<typename T> __global__ void
pooling_max_backward_kernel
(T* derData,
 const T* data,
 const T* derOutput,
 const int outputWidth,
 const int outputHeight,
 const int outputVolume,
 const int width,
 const int height,
 const int poolWidth,
 const int poolHeight,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  int outputIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (outputIndex < outputVolume) {
    int px = outputIndex ;
    int py = px / outputWidth ;
    int pz = py / outputHeight ;
    px %= outputWidth ;
    py %= outputHeight ;
    data += pz * (width*height) ;
    derData += pz * (width*height) ;

    int x1 = px * strideX - padLeft ;
    int y1 = py * strideY - padTop ;
    int x2 = min(x1 + poolWidth, width) ;
    int y2 = min(y1 + poolHeight, height) ;
    x1 = max(x1, 0) ;
    y1 = max(y1, 0) ;

    int bestIndex = y1 * width + x1 ;
    T bestValue = data[bestIndex] ;
    for (int y = y1 ; y < y2 ; ++y) {
      for (int x = x1 ; x < x2 ; ++x) {
        int index = y * width + x ;
        T value = data[index] ;
        if (value > bestValue) {
          bestValue = value ;
          bestIndex = index ;
        }
      }
    }
    /*
     This is bad, but required to eliminate a race condition when writing
     to bottom_diff.
     Caffe goes the other way around, but requrires remembering the layer
     output, or the maximal indexes.
     atomicAdd(add, val)
     */
    atomicAdd(derData + bestIndex, derOutput[outputIndex]) ;
  }
}

// -------------------------------------------------------------------
//                                             Average pooling helpers
// -------------------------------------------------------------------

template<typename T> __global__ void
pooling_average_kernel
(T* output,
 const T* data,
 const int outputWidth,
 const int outputHeight,
 const int outputVolume,
 const int width,
 const int height,
 const int poolWidth,
 const int poolHeight,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  /* outputIndex = x + y * outputWidth + z * (outputWidth * outputHeight) */
  int outputIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (outputIndex < outputVolume) {
    int px = outputIndex ;
    int py = px / outputWidth ;
    int pz = py / outputHeight ;
    px %= outputWidth ;
    py %= outputHeight ;
    int x1 = px * strideX - padLeft ;
    int y1 = py * strideY - padTop ;
    int x2 = min(x1 + poolWidth, width) ;
    int y2 = min(y1 + poolHeight, height) ;
    x1 = max(x1, 0) ;
    y1 = max(y1, 0) ;
    data += pz * (width*height) ;
    T accum = 0;
    T poolSize = (y2 - y1)*(x2 - x1);
    for (int y = y1 ; y < y2 ; ++y) {
      for (int x = x1 ; x < x2 ; ++x) {
        accum += data[y * width + x] ;
      }
    }
    output[outputIndex] = accum / poolSize ;
  }
}

template <typename T> __global__ void
pooling_average_backward_kernel
(T* derData,
 const T* derOutput,
 const int nthreads,
 const int outputWidth,
 const int outputHeight,
 const int width,
 const int height,
 const int depth,
 const int poolWidth,
 const int poolHeight,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    /* To understand the logic of this piece of code see the
     comments to of the row2im backward kernel */
    int x_data = index ;
    int y_data = x_data / width ;
    int z = y_data / height ;
    x_data %= width ;
    y_data %= height ;

    int dx = x_data + padLeft - poolWidth ;
    int dy = y_data + padTop - poolHeight ;
    int px1 = (dx >= 0) ? dx/strideX + 1 : 0 ;
    int py1 = (dy >= 0) ? dy/strideY + 1 : 0 ;
    int px2 = min((x_data + padLeft) / strideX, outputWidth - 1) ;
    int py2 = min((y_data + padTop) / strideY, outputHeight - 1) ;
    T accumulator = 0 ;
    derOutput += z * outputHeight * outputWidth;
    for (int py = py1 ; py <= py2 ; ++py) {
      for (int px = px1 ; px <= px2 ; ++px) {
        int x1 = px * strideX - padLeft ;
        int y1 = py * strideY - padTop ;
        int x2 = min(x1 + poolWidth, width) ;
        int y2 = min(y1 + poolHeight, height) ;
        x1 = max(x1, 0) ;
        y1 = max(y1, 0) ;
        T poolSize = (y2 - y1) * (x2 - x1);
        accumulator += derOutput[py * outputWidth + px] / poolSize ;
      }
    }
    derData[index] = accumulator ;
  }
}

// -------------------------------------------------------------------
//                                                             Forward
// -------------------------------------------------------------------

template<DataType dataType, Pooling::Method method>
struct PoolingForwardGPU
{
  vl::ErrorCode operator()(Pooling &op,
                           Tensor &output,
                           Tensor const &input)
  {
    typedef typename vl::DataTypeTraits<dataType>::type type ;
    auto height = input.getHeight() ;
    auto width = input.getWidth() ;
    auto depth = input.getDepth() ;
    auto size = input.getSize() ;
    auto inputData = (type const*)input.getMemory() ;
    auto outputData = (type*)output.getMemory() ;
    auto outputWidth = (width + (op.padLeft + op.padRight) - op.poolWidth)/op.strideX + 1 ;
    auto outputHeight = (height + (op.padTop + op.padBottom) - op.poolHeight)/op.strideY + 1 ;
    auto outputVolume = outputWidth * outputHeight * depth * size ;

    if (method == Pooling::Max) {
      pooling_max_kernel<type>
      <<< divideAndRoundUp(outputVolume, (size_t)VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
      (outputData, inputData,
       outputHeight, outputWidth, outputVolume,
       height, width,
       op.poolHeight, op.poolWidth,
       op.strideY, op.strideX,
       op.padTop, op.padLeft);
    }
    else if (method == Pooling::Average) {
      pooling_average_kernel<type>
      <<< divideAndRoundUp(outputVolume, (size_t)VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
      (outputData, inputData,
       outputHeight, outputWidth, outputVolume,
       height, width,
       op.poolHeight, op.poolWidth,
       op.strideY, op.strideX,
       op.padTop, op.padLeft);
    }
    else {
      assert(false) ;
    }

    hipError_t status = hipPeekAtLastError() ;
    return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
  }
} ;

template<DataType dataType>
struct PoolingForward<VLDT_GPU,dataType>
{
  vl::ErrorCode operator()(Pooling &op,
                           Tensor output,
                           Tensor input)
  {
    switch (op.method) {
      case Pooling::Max:
        return
        PoolingForwardGPU<dataType,Pooling::Max>
        ()(op,output,input) ;
      case Pooling::Average:
        return
        PoolingForwardGPU<dataType,Pooling::Average>
        ()(op,output,input) ;
      default:
        return VLE_IllegalArgument ;
    }
  }
} ;

// -------------------------------------------------------------------
//                                                            Backward
// -------------------------------------------------------------------

template<DataType dataType, Pooling::Method method>
struct PoolingBackwardGPU
{
  vl::ErrorCode operator()(Pooling &op,
                           Tensor &derInput,
                           Tensor const &input,
                           Tensor const &derOutput)
  {
    typedef typename vl::DataTypeTraits<dataType>::type type ;
    auto height = input.getHeight() ;
    auto width = input.getWidth() ;
    auto depth = input.getDepth() ;
    auto size = input.getSize() ;
    auto inputData = (type const*)input.getMemory() ;
    auto derOutputData = (type const*)derOutput.getMemory() ;
    auto derInputData = (type*)derInput.getMemory() ;
    auto outputWidth = (width + (op.padLeft + op.padRight) - op.poolWidth)/op.strideX + 1 ;
    auto outputHeight = (height + (op.padTop + op.padBottom) - op.poolHeight)/op.strideY + 1 ;
    auto outputVolume = outputWidth * outputHeight * depth * size ;
    auto inputVolume = width * height * size * depth ;

    if (method == Pooling::Max) {
      pooling_max_backward_kernel<type>
      <<< divideAndRoundUp(outputVolume, (size_t)VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
      (derInputData, inputData, derOutputData,
       outputHeight, outputWidth, outputVolume,
       height, width,
       op.poolHeight, op.poolWidth,
       op.strideY, op.strideX,
       op.padTop, op.padLeft);
    }
    else if (method == Pooling::Average) {
      pooling_average_backward_kernel<type>
      <<< divideAndRoundUp(inputVolume, (size_t)VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
      (derInputData, derOutputData, inputVolume,
       outputHeight, outputWidth,
       height, width, size * depth,
       op.poolHeight, op.poolWidth,
       op.strideY, op.strideX,
       op.padTop, op.padLeft) ;
    }
    else {
      assert(false) ;
    }

    hipError_t status = hipPeekAtLastError() ;
    return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
  }
} ; // pooling_max


template<DataType dataType>
struct PoolingBackward<VLDT_GPU,dataType>
{
  vl::ErrorCode operator()(Pooling &op,
                           Tensor &derInput,
                           Tensor const &input,
                           Tensor const &derOutput)
  {
    switch (op.method) {
      case Pooling::Max:
        return
        PoolingBackwardGPU<dataType,Pooling::Max>
        ()(op,derInput,input,derOutput) ;
      case Pooling::Average:
        return
        PoolingBackwardGPU<dataType,Pooling::Average>
        ()(op,derInput,input,derOutput) ;
      default:
        return VLE_IllegalArgument ;
    }
  }
} ;
