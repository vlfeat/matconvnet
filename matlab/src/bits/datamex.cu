// @file datamex.cu
// @brief Basic data structures (MEX support)
// @author Andrea Vedaldi

/*
Copyright (C) 2015-16 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "datamex.hpp"
#if ENABLE_GPU
#include "datacu.hpp"
#endif

#ifndef NDEBUG
#include<iostream>
#endif

#include "impl/copy.hpp"

using namespace vl ;
using namespace vl::impl ;

/* ---------------------------------------------------------------- */
/*                                                       MexContext */
/* ---------------------------------------------------------------- */

vl::MexContext::MexContext()
: Context()
#if ENABLE_GPU
, gpuIsInitialized(false)
, canary(NULL)
#endif
{ }

vl::MexContext::~MexContext()
{
#if ENABLE_GPU
  // so that ~Context does not crash if MATLAB resets the GPU in the mean time
  validateGpu() ;
#endif
}

/* ---------------------------------------------------------------- */
/*                                                   GPU management */
/* ---------------------------------------------------------------- */

#if ENABLE_GPU

// Do noting if the GPU is not initialized, otherwise invalidate it
// if needed
vl::Error
MexContext::validateGpu()
{
  if (!gpuIsInitialized) { return vl::vlSuccess ; }
  gpuIsInitialized = mxGPUIsValidGPUData(canary) ;
  if (!gpuIsInitialized) {
#ifndef NDEBUG
    std::cout<<"MexContext:: GPU reset detected; invalidating the GPU state"<<std::endl ;
#endif
    mxDestroyArray(canary) ;
    canary = NULL ;
    Context::invalidateGpu() ;
  }
  return vl::vlSuccess ;
}

// Initialize GPU; also make sure that it was not reset by MATLAB
vl::Error
vl::MexContext::initGpu()
{
  validateGpu() ;
  if (!gpuIsInitialized) {
    mwSize dims = 1 ;
    mxInitGPU() ;
    // todo: can mxGPUCreateGPUArray return NULL ?
    mxGPUArray * gpuArray =
    mxGPUCreateGPUArray(1,&dims,mxINT8_CLASS,mxREAL,MX_GPU_DO_NOT_INITIALIZE) ;
    canary = mxGPUCreateMxArrayOnGPU(gpuArray) ;
    mexMakeArrayPersistent(canary) ;
    mxGPUDestroyGPUArray(gpuArray) ;
    gpuIsInitialized = true ;
  }
  return vl::vlSuccess ;
}
#endif

/* ---------------------------------------------------------------- */
/*                                                        MexTensor */
/* ---------------------------------------------------------------- */

/*
 The MexTensor class helps handling MATLAB CPU and GPU arrays.

 The design is somewhat ackward to match MATLAB assumpitons.

 The class can either:

 - wrap an existing mxArray (or mxArray + mxGPUArray)
 - or create a new mxArray (or mxArray + mxGPUArray)

 In the last case, the array is released when the destructor is
 called. However, this would normally interfere with MATLAB
 automatic garbage collection upon raising an exception (which
 can happen using mexErrMsgTxt() or, implicitly, when an array
 creation function cannot complete, for example due to a memory error).

 Therefore the constructors make the allocated memory persistent. C++
 guarantees that the arrays are freeed upon error in the destructors.

 Note that, upon cerating an array, errors such as running out of
 CPU/GPU memory can occurr. In this case, MATLAB throws an error
 and quits the MEX file (either implicitly or because we call
 mexErrMsgTxt()). Hence constructors always complete with a well
 defined object.

 */

/* ---------------------------------------------------------------- */
/* Constructing, clearing, destroying                               */
/* ---------------------------------------------------------------- */

vl::MexTensor::MexTensor(MexContext & context)
: context(context),
Tensor(),
array(NULL),
isArrayOwner(false)
#if ENABLE_GPU
, gpuArray(NULL)
#endif
{ }

mxArray *
vl::MexTensor::relinquish()
{
  if (isArrayOwner) {
    isArrayOwner = false ;
    return (mxArray*) array ;
  } else {
    // this is because we may be encapsulating an input argument
    // and we may be trying to return it
    // we should probably use the undocumented
    // extern mxArray *mxCreateSharedDataCopy(const mxArray *pr);
    return mxDuplicateArray(array) ;
  }
}

void
vl::MexTensor::clear()
{
#if ENABLE_GPU
  if (gpuArray) {
    mxGPUDestroyGPUArray(gpuArray) ;
    gpuArray = NULL ;
  }
#endif
  if (isArrayOwner) {
    if (array) {
      mxDestroyArray((mxArray*)array) ;
      array = NULL ;
    }
    isArrayOwner = false ;
  }
  memory = NULL ;
  memorySize = 0 ;
  deviceType = vl::CPU ;
  vl::TensorShape::clear() ;
}

vl::MexTensor::~MexTensor()
{
  clear() ;
}

/* ---------------------------------------------------------------- */
/* init with optional zero filling                                  */
/* ---------------------------------------------------------------- */


vl::Error
vl::MexTensor::initHelper(Device newDeviceType, Type newDataType,
                          TensorShape const & newShape, bool fillWithZeros)
{
  clear() ;

  // assign dimensions
  mwSize dimensions [VL_TENSOR_SHAPE_MAX_NUM_DIMENSIONS] ;
  for (int k = 0 ; k < newShape.getNumDimensions() ; ++k) {
    dimensions[k] = (mwSize)newShape.getDimension(k) ;
  }

  // compute the size in bytes
  mwSize newMemorySize = newShape.getNumElements() ;
  mxClassID classID ;
  switch (newDataType) {
    case vlTypeFloat:
      newMemorySize *= sizeof(DataTypeTraits<vlTypeFloat>::type) ;
      classID = mxSINGLE_CLASS ;
      break ;
#ifdef ENABLE_DOUBLE
    case vlTypeDouble:
      newMemorySize *= sizeof(DataTypeTraits<vlTypeDouble>::type) ;
      classID = mxDOUBLE_CLASS ;
      break ;
#endif
    default:
      abort() ;
  }

  // allocate the memory on CPU or GPU
  void * newMemory = NULL ;
  mxArray * newArray = NULL ;
#if ENABLE_GPU
  mxGPUArray* newGpuArray = NULL ;
#endif

  if (newDeviceType == vl::CPU) {
    if (fillWithZeros) {
      newArray = mxCreateNumericArray(4, dimensions, classID, mxREAL) ;
      newMemory = mxGetData(newArray) ;
    } else {
      mwSize dimensions_ [1] = {0} ;
      newMemory = mxMalloc(newMemorySize) ;
      newArray = mxCreateNumericArray(1, dimensions_,
                                      classID,
                                      mxREAL) ;
      mxSetData(newArray, newMemory) ;
      mxSetDimensions(newArray, dimensions, newShape.getNumDimensions()) ;
    }
  }
#ifdef ENABLE_GPU
  else {
    context.initGpu() ;
    newGpuArray = mxGPUCreateGPUArray(newShape.getNumDimensions(), dimensions,
                                      classID,
                                      mxREAL,
                                      fillWithZeros ? MX_GPU_INITIALIZE_VALUES : MX_GPU_DO_NOT_INITIALIZE) ;
    newArray = mxGPUCreateMxArrayOnGPU(newGpuArray) ;
    newMemory = mxGPUGetData(newGpuArray) ;
  }
#else
  else {
    abort() ;
  }
#endif

  //mexMakeArrayPersistent(newArray) ; // avoid double free with MATALB garbage collector upon error
  TensorShape::operator=(newShape) ;
  deviceType = newDeviceType ;
  dataType = newDataType ;
  memory = newMemory ;
  memorySize = newMemorySize ;
  array = newArray ;
  isArrayOwner = true ;
#if ENABLE_GPU
  gpuArray = newGpuArray ;
#endif
  return vl::vlSuccess ;
}

/* ---------------------------------------------------------------- */
/*                                                          init*() */
/* ---------------------------------------------------------------- */

vl::Error
vl::MexTensor::init(Device newDeviceType,
                    Type newDataType,
                    TensorShape const & newShape)
{
  return initHelper(newDeviceType, newDataType, newShape, false) ;
}

vl::Error
vl::MexTensor::initWithZeros(Device newDeviceType,
                             Type newDataType,
                             TensorShape const & newShape)
{
  return initHelper(newDeviceType, newDataType, newShape, true) ;
}

vl::Error
vl::MexTensor::initWithValue(Device newDeviceType,
                             Type newDataType,
                             TensorShape const & newShape,
                             double value)
{
  if (value == 0) {
    return initHelper(newDeviceType, newDataType, newShape, true) ;
  } else {
    vl::Error error = initHelper(newDeviceType, newDataType, newShape, false) ;
    if (error != vlSuccess) { return error ; }
    size_t const n = getNumElements() ;
    if (newDeviceType == vl::CPU) {
      switch (newDataType) {
        case vlTypeFloat: error = operations<vl::CPU,float>::fill((float*)memory, n, (float)value) ; break ;
#ifdef ENABLE_DOUBLE
        case vlTypeDouble: error = operations<vl::CPU,double>::fill((double*)memory, n, (double)value) ; break ;
#endif
        default: abort() ;
      }
    }
#ifdef ENABLE_GPU
    else {
      switch (newDataType) {
        case vlTypeFloat: error = operations<vl::GPU,float>::fill((float*)memory, n, (float)value) ; break ;
#ifdef ENABLE_DOUBLE
        case vlTypeDouble: error = operations<vl::GPU,double>::fill((double*)memory, n, (double)value) ; break ;
#endif
        default: abort() ;
      }
      if (error == vlErrorCuda) {
        hipError_t error = hipGetLastError() ;
        clear() ;
        mexErrMsgTxt((std::string("MexTensor: fill [CUDA error: ")
                      + hipGetErrorString(error)
                      + "]"
                      ).c_str()) ;
      }
    }
#endif
  }
  return vl::vlSuccess ;
}

/* ---------------------------------------------------------------- */
/* init by wrapping a given array                                   */
/* ---------------------------------------------------------------- */

vl::Error
vl::MexTensor::init(mxArray const * array_)
{
  clear() ;
  if (array_ == NULL) { return vl::vlSuccess ; } // empty

  vl::Device newDeviceType ;
  vl::Type newDataType ;
  void const * newMemory = NULL ;
  mxArray * newArray = (mxArray*)array_ ;
#if ENABLE_GPU
  mxGPUArray* newGpuArray = NULL ;
#endif

  mwSize const * newDimensions ;
  mwSize newNumDimensions ;
  mxClassID newClassID ;

#ifdef ENABLE_GPU
  context.initGpu() ;
  if (mxIsGPUArray(array_)) {
    newDeviceType = GPU ;
    newGpuArray = (mxGPUArray*) mxGPUCreateFromMxArray(newArray) ;
    newMemory = mxGPUGetDataReadOnly(newGpuArray) ;
    newClassID = mxGPUGetClassID(newGpuArray) ;
    newDimensions = mxGPUGetDimensions(newGpuArray) ;
    newNumDimensions = mxGPUGetNumberOfDimensions(newGpuArray) ;
  } else
#endif
  {
    if (!mxIsNumeric(newArray)) {
      mexErrMsgTxt("An input is not a numeric array (or GPU support not compiled).") ;
    }
    newDeviceType = CPU ;
    newMemory = mxGetData(newArray) ;
    newClassID = mxGetClassID(newArray) ;
    newDimensions = mxGetDimensions(newArray) ;
    newNumDimensions = mxGetNumberOfDimensions(newArray) ;
  }

  if (newNumDimensions >= VL_TENSOR_SHAPE_MAX_NUM_DIMENSIONS) {
#if ENABLE_GPU
    if (newGpuArray) {
      mxGPUDestroyGPUArray(newGpuArray) ;
      newGpuArray = NULL ;
    }
#endif
    mexErrMsgTxt("An input has more than the maximum number of allowed dimensions.") ;
  }

  numDimensions = newNumDimensions ;
  for (int k = 0 ; k < numDimensions ; ++k) {
    setDimension(k, newDimensions[k]) ;
  }

  size_t newMemorySize = getNumElements() ;

  switch (newClassID) {
    case mxSINGLE_CLASS:
      newDataType = vlTypeFloat ;
      newMemorySize *= sizeof(DataTypeTraits<vlTypeFloat>::type) ;
      break ;

#ifdef ENABLE_DOUBLE
    case mxDOUBLE_CLASS:
      newDataType = vlTypeDouble ;
      newMemorySize *= sizeof(DataTypeTraits<vlTypeDouble>::type) ;
      break ;
#endif

    default:
      if (isEmpty()) {
        newDataType = vlTypeFloat ;
        newMemorySize = 0 ;
        break ;
      }
#ifdef ENABLE_DOUBLE
      mexErrMsgTxt("An input is neither SINGLE or DOUBLE nor it is empty.") ;
#else
      mexErrMsgTxt("An input is neither SINGLE nor empty.") ;
#endif
      break ;
  }

  deviceType = newDeviceType ;
  dataType = newDataType ;
  memory = (void*)newMemory ;
  memorySize = newMemorySize ;
  array = newArray ;
  isArrayOwner = false ;
#if ENABLE_GPU
  gpuArray = newGpuArray ;
#endif


  return vl::vlSuccess ;
}

size_t
vl::MexTensor::getMemorySize() const
{
  return memorySize ;
}

void vl::print(char const * str, vl::MexTensor const & tensor)
{
  size_t size = tensor.getMemorySize() ;
  double scaled ;
  size_t const * dimensions = tensor.getDimensions() ;
  const char * units ;
  const char * type ;
  if (size < 1024) {
    scaled = size ;
    units = "B" ;
  } else if (size < 1024*1024) {
    scaled = size / 1024.0 ;
    units = "KB" ;
  } else if (size < 1024*1024*1024) {
    scaled = size / (1024.0 * 1024.0) ;
    units = "MB" ;
  } else {
    scaled = size / (1024.0 * 1024.0 * 1024.0) ;
    units = "GB" ;
  }
  const char * dev = "" ;
  switch (tensor.getDeviceType()) {
    case vl::CPU : dev = "CPU" ; break ;
    case vl::GPU : dev = "GPU" ; break ;
  }
  switch (tensor.getDataType()) {
    case vl::vlTypeFloat: type = "float" ; break ;
    case vl::vlTypeDouble: type = "double" ; break ;
    case vl::vlTypeChar: type = "char" ; break ;
    default: type = "uknown type" ;
  }
  mexPrintf("%s[", str) ;
  for (int k = 0 ; k < tensor.getNumDimensions() ; ++k) {
    mexPrintf("%d ", dimensions[k]) ;
  }
  mexPrintf("| %s %.1f%s %s]\n",
            type,
            scaled,
            units,
            dev);
}
