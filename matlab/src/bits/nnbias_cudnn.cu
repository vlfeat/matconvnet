// @file nnbias_cudnn.cu
// @brief biasolution block CuDNN-based implementation.
// @author Andrea Vedaldi

/*
Copyright (C) 2015-17 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnbias.hpp"
#include "datacu.hpp"
#include "impl/cudnnhelper.hpp"
#include <cassert>

using namespace std ;
using namespace vl ;
using namespace vl::nn ;
using namespace vl::impl ;

#define CHECK(x) \
{ \
cudnnError = x ; \
if (cudnnError != HIPDNN_STATUS_SUCCESS) { \
error = op.context.setError(op.context.getCudaHelper().catchCudnnError(cudnnError, \
STRINGIZE(__LINE__) ":" STRINGIZE(__FILE__))) ; \
goto done ; \
} }

// -------------------------------------------------------------------
//                                                             Forward
// -------------------------------------------------------------------

template<DataType dataType>
struct BiasForwardCudnn
{
  vl::ErrorCode operator()(Bias & op,
                           Tensor &output, double outputMult,
                           Tensor const &input, double inputMult,
                           Tensor const &bias, double biasMult)
  {
    typedef typename DataTypeTraits<dataType>::type type ;

    hipdnnTensorDescriptor_t outputDesc, biasDesc, dataDesc ;
    bool outputDescInitialized = false ;
    bool biasDescInitialized = false ;
    bool dataDescInitialized = false ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN
    CHECK(op.context.getCudaHelper().getCudnnHandle(&handle)) ;

    // Get output tensor descripotr
    assert(output) ;
    CHECK(hipdnnCreateTensorDescriptor(&outputDesc)) ;
    outputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(outputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     DataTypeToCudnn<dataType>::dataType,
                                     output.getSize(), // sizes
                                     output.getDepth(),
                                     output.getWidth(),
                                     output.getHeight())) ;

    if (bias) {
      CHECK(hipdnnCreateTensorDescriptor(&biasDesc)) ;
      biasDescInitialized = true ;
      CHECK(hipdnnSetTensor4dDescriptor(biasDesc,
                                       HIPDNN_TENSOR_NCHW,
                                       DataTypeToCudnn<dataType>::dataType,
                                       1,
                                       bias.getNumElements(),
                                       1,
                                       1)) ;

      type alpha = biasMult ;
      type beta = outputMult ;
#if (CUDNN_VERSION < 4000)
      CHECK(hipdnnAddTensor(handle,
                           CUDNN_ADD_SAME_C,
                           &alpha,
                           biasDesc, bias.getMemory(),
                           &beta,
                           outputDesc, output.getMemory())) ;
#else
      CHECK(hipdnnAddTensor(handle,
                           &alpha,
                           biasDesc, bias.getMemory(),
                           &beta,
                           outputDesc, output.getMemory())) ;
#endif
      outputMult = 1 ;
    }

    if (input) {
      CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
      dataDescInitialized = true ;
      CHECK(hipdnnSetTensor4dDescriptor(dataDesc,
                                       HIPDNN_TENSOR_NCHW,
                                       DataTypeToCudnn<dataType>::dataType,
                                       input.getSize(),
                                       input.getDepth(),
                                       input.getWidth(),
                                       input.getHeight())) ;

      type alpha = inputMult ;
      type beta = outputMult ;
#if (CUDNN_VERSION < 4000)
      CHECK(hipdnnAddTensor(handle,
                           CUDNN_ADD_FULL_TENSOR,
                           &alpha,
                           dataDesc, input.getMemory(),
                           &beta,
                           outputDesc, output.getMemory()));
#else
      CHECK(hipdnnAddTensor(handle,
                           &alpha,
                           dataDesc, input.getMemory(),
                           &beta,
                           outputDesc, output.getMemory()));
#endif
    }

    /* cleanup */
  done:
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    if (biasDescInitialized) { hipdnnDestroyTensorDescriptor(biasDesc) ; }
    if (outputDescInitialized) { hipdnnDestroyTensorDescriptor(outputDesc) ; }
    return op.context.passError(error, __func__) ;
  }
} ;


// -------------------------------------------------------------------
//                                                            Backward
// -------------------------------------------------------------------

template<DataType dataType>
struct BiasBackwardCudnn
{
  vl::ErrorCode operator()(Bias &op,
                           Tensor &derInput, double derInputMult,
                           Tensor &derBias, double derBiasMult,
                           double inputMult, double biasMult,
                           Tensor const &derOutput)
  {
    typedef typename DataTypeTraits<dataType>::type type ;

    /* no derInputDesc needed as same as dataDesc */
    hipdnnTensorDescriptor_t derInputDesc, derBiasDesc, derOutputDesc ;
    bool derInputDescInitialized = false ;
    bool derBiasDescInitialized = false ;
    bool derOutputDescInitialized = false ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN
    CHECK(op.context.getCudaHelper().getCudnnHandle(&handle)) ;

    // Must have derOutput for all derivatives
    assert(derOutput) ;
    CHECK(hipdnnCreateTensorDescriptor(&derOutputDesc)) ;
    derOutputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(derOutputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     DataTypeToCudnn<dataType>::dataType,
                                     derOutput.getSize(), // sizes
                                     derOutput.getDepth(),
                                     derOutput.getWidth(),
                                     derOutput.getHeight())) ;

    // for derivatives w.r.t. bias
    if (derBias) {
      CHECK(hipdnnCreateTensorDescriptor(&derBiasDesc)) ;
      derBiasDescInitialized = true ;
      CHECK(hipdnnSetTensor4dDescriptor(derBiasDesc,
                                       HIPDNN_TENSOR_NCHW,
                                       DataTypeToCudnn<dataType>::dataType,
                                       1,
                                       derBias.getNumElements(),
                                       1,
                                       1)) ;

      type alpha = biasMult ;
      type beta = derBiasMult ;
      CHECK(hipdnnConvolutionBackwardBias
            (handle,
             &alpha,
             derOutputDesc, (type const*)derOutput.getMemory(),
             &beta,
             derBiasDesc, (type*)derBias.getMemory())) ;
    }

    if (derInput) {
      CHECK(hipdnnCreateTensorDescriptor(&derInputDesc)) ;
      derInputDescInitialized = true ;
      CHECK(hipdnnSetTensor4dDescriptor(derInputDesc,
                                       HIPDNN_TENSOR_NCHW,
                                       DataTypeToCudnn<dataType>::dataType,
                                       derInput.getSize(),
                                       derInput.getDepth(),
                                       derInput.getWidth(),
                                       derInput.getHeight())) ;
      type alpha = inputMult ;
      type beta = derInputMult ;
#if (CUDNN_VERSION < 4000)
      CHECK(hipdnnAddTensor(handle,
                           CUDNN_ADD_SAME_C,
                           &alpha,
                           biasDesc, bias.getMemory(),
                           &beta,
                           derInputDesc, derInput.getMemory())) ;
#else
      CHECK(hipdnnAddTensor(handle,
                           &alpha,
                           derOutputDesc, derOutput.getMemory(),
                           &beta,
                           derInputDesc, derInput.getMemory())) ;
#endif
    }

  done:
    if (derOutputDescInitialized) { hipdnnDestroyTensorDescriptor(derOutputDesc) ; }
    if (derBiasDescInitialized) { hipdnnDestroyTensorDescriptor(derBiasDesc) ; }
    if (derInputDescInitialized) { hipdnnDestroyTensorDescriptor(derInputDesc) ; }
    return op.context.passError(error, __func__) ;
  }
} ;
