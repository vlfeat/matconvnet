#include "hip/hip_runtime.h"
// @file nnbilinearsampler.cu
// @brief Bilinear sampler block
// @author Ankush Gupta
// @author Andrea Vedaldi

/*
Copyright (C) 2016-17 Ankush Gupta and Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnbilinearsampler.hpp"
#include "impl/dispatcher.hpp"
#include <cassert>
#include <cmath>

using namespace vl ;
using namespace vl::nn ;
using namespace vl::impl ;

template<DeviceType deviceType, DataType dataType> struct BilinearSamplerForward ;
template<DeviceType deviceType, DataType dataType> struct BilinearSamplerBackward ;
template<DataType dataType> struct BilinearSamplerForwardCudnn ;
template<DataType dataType> struct BilinearSamplerBackwardCudnn ;

// -------------------------------------------------------------------
//                                                             Helpers
// -------------------------------------------------------------------

template<typename type, bool backwardData, bool backwardGrid>
static vl::ErrorCode
forward_backward
(vl::Context& context,
 type* output,
 type* derData,
 type* derGrid,
 type const* data,
 type const* grid,
 type const* derOutput,
 size_t outHeight, size_t outWidth, size_t outDepth, size_t outCardinality,
 size_t inHeight, size_t inWidth, size_t inCardinality)
{
  vl::ErrorCode error = vl::VLE_Success ;

  bool backward = backwardData | backwardGrid ;

  // common conditions
  assert(grid) ;
  assert(divides(inCardinality, outCardinality)) ;

  // forward conditions
  assert(backward || data) ;
  assert(backward || output) ;

  // backward conditions
  assert(!backward || derOutput) ;
  assert(!backwardData || derData) ;
  assert(!backwardGrid || derGrid) ;
  assert(!backwardGrid || data) ;

  int groupSize = outCardinality / inCardinality ;

  // don't need these -- as already being initialized with zeros in the mex file:
  // if (backwardData) {
  //   memset(derData, 0, inHeight * inWidth * outDepth * inCardinality * sizeof(type)) ;
  // }
  // if (backwardGrid) {
  //   memset(derGrid, 0, 2 * outHeight * outWidth * outCardinality * sizeof(type)) ;
  // }
  for (int n = 0 ; n < outCardinality ; ++n) {
    for (int c = 0 ; c < outDepth ; ++c) {
      type const * end = grid + 2 * outWidth * outHeight ;
      while (grid < end) {
        type py = *grid++ ;
        type px = *grid++ ;

        py = type(0.5)*(py + type(1.0)) * (inHeight - 1) ;
        px = type(0.5)*(px + type(1.0)) * (inWidth - 1) ;
        const int sx = floor(px); // todo: check floor vs floorf
        const int sy = floor(py);

        type acc = 0 ;
        type dgridx = 0 ;
        type dgridy = 0 ;
        type dy ;
        if (backward) {
          dy = *derOutput++ ;
        }

        // todo: check boundary conditions in other frameworks and make
        // them the same
        if (-1 <= sy && sy < inHeight && -1 <= sx && sx < inWidth) {
          // get the interpolation weights
          const type wx = px - sx ;
          const type wy = py - sy ;

#pragma unroll
          for (int j=0; j < 2; j++) {
#pragma unroll
            for (int i=0; i < 2; i++) {
              int ssy = sy + i ;
              int ssx = sx + j ;
              if (ssy < 0 || ssy >= inHeight || ssx < 0 || ssx >= inWidth) {
                continue ;
              }
              type wwx = (1-j)*(1-wx) + j*wx ;
              type wwy = (1-i)*(1-wy) + i*wy ;
              type ww = wwx * wwy ;
              if (!backward) {
                acc += ww * data[ssy + ssx * inHeight];
              } else {
                if (backwardData) {
                  derData[ssy + ssx * inHeight] += ww * dy ;
                }
                if (backwardGrid) {
                  type x = data[ssy + ssx * inHeight] ;
                  dgridx += (2*j-1) * wwy * dy * x ;
                  dgridy += (2*i-1) * wwx * dy * x ;
                }
              }
            }
          }
        }
        if (!backward) {
          *output++ = acc ;
        }
        if (backwardGrid) {
          *derGrid++ += type(0.5)*(inHeight - 1) * dgridy ;
          *derGrid++ += type(0.5)*(inWidth - 1) * dgridx ;
        }
      }
      // next channel
      data += inHeight * inWidth ;
      derData +=inHeight * inWidth ;
      grid -= 2 * outHeight * outWidth ;
      derGrid -= 2 * outHeight * outWidth ;
    }
    // next image
    if ((n + 1) % groupSize != 0) {
      data -= inHeight * inWidth * outDepth ;
      derData -= inHeight * inWidth * outDepth ;
    }
    grid += 2 * outHeight * outWidth ;
    derGrid += 2 * outHeight * outWidth ;
  }
  return error ;
}

// -------------------------------------------------------------------
//                                                             Forward
// -------------------------------------------------------------------

template<DataType dataType>
struct BilinearSamplerForward<VLDT_CPU,dataType>
{
  vl::ErrorCode operator()
  (BilinearSampler &op,
   Tensor &output,
   Tensor const &input,
   Tensor const &grid)
  {
    typedef typename DataTypeTraits<dataType>::type type ;

    auto outHeight = output.getHeight() ;
    auto outWidth = output.getWidth() ;
    auto outDepth = output.getDepth() ;
    auto outCardinality = output.getSize() ;
    auto inHeight = input.getHeight() ;
    auto inWidth = input.getWidth() ;
    auto inCardinality = input.getSize() ;
    auto outputData = (type*)output.getMemory() ;
    auto inputData = (type const*)input.getMemory() ;
    auto gridData = (type const*)grid.getMemory() ;

    return forward_backward<type, false, false>
    (op.context, outputData, NULL, NULL, inputData, gridData, NULL,
     outHeight, outWidth, outDepth, outCardinality,
     inHeight, inWidth, inCardinality) ;
  }
} ;

// -------------------------------------------------------------------
//                                                            Backward
// -------------------------------------------------------------------

#define DISPATCH(bwData, bwGrid) \
error = forward_backward<type, bwData, bwGrid> \
(op.context, NULL, derInputData, derGridData, inputData, gridData, derOutputData, \
outHeight, outWidth, outDepth, outCardinality, \
inHeight, inWidth,inCardinality) ;

template<DataType dataType>
struct BilinearSamplerBackward<VLDT_CPU,dataType>
{
  vl::ErrorCode operator()
  (BilinearSampler &op,
   Tensor &derInput,
   Tensor &derGrid,
   Tensor const &input,
   Tensor const &grid,
   Tensor const &derOutput)
  {
    typedef typename DataTypeTraits<dataType>::type type ;
    auto outHeight = derOutput.getHeight() ;
    auto outWidth = derOutput.getWidth() ;
    auto outDepth = derOutput.getDepth() ;
    auto outCardinality = derOutput.getSize() ;
    auto inHeight = input.getHeight() ;
    auto inWidth = input.getWidth() ;
    auto inCardinality = input.getSize() ;
    auto derInputData = (type*)derInput.getMemory() ;
    auto derGridData = (type*)derGrid.getMemory() ;
    auto inputData = (type const*)input.getMemory() ;
    auto gridData = (type const*)grid.getMemory() ;
    auto derOutputData = (type const*)derOutput.getMemory() ;

    vl::ErrorCode error = VLE_Success ;

    // optimized codepaths depending on what needs to be comptued
    if (derInput && !derGrid) {
      DISPATCH(true, false) ;
    } else if (!derInputData && derGridData) {
      DISPATCH(false, true) ;
    } else if (derInputData && derGridData) {
      DISPATCH(true, true) ;
    }
    return error ;
  }
} ;

// -------------------------------------------------------------------
//                                                             Drivers
// -------------------------------------------------------------------

#if ENABLE_GPU
#include "nnbilinearsampler_gpu.cu"
#endif

#if ENABLE_CUDNN
#include "nnbilinearsampler_cudnn.cu"
#endif

BilinearSampler::BilinearSampler(Context &context)
: context(context)
{ }

vl::ErrorCode
BilinearSampler::forward(Tensor &output,
                         Tensor const &input,
                         Tensor const &grid)
{
  return dispatch_cudnn<
  BilinearSamplerForward,
  BilinearSamplerForwardCudnn>()
  (*this,output,input,grid) ;
}

vl::ErrorCode
BilinearSampler::backward(Tensor &derInput,
                          Tensor &derGrid,
                          Tensor const &input,
                          Tensor const &grid,
                          Tensor const &derOutput)
{
  return dispatch_cudnn<
  BilinearSamplerBackward,
  BilinearSamplerBackwardCudnn>()
  (*this,derInput,derGrid,input,grid,derOutput) ;
}
