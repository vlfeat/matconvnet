#include "hip/hip_runtime.h"
// @file nnpooling.cu
// @brief Pooling block
// @author Andrea Vedaldi

/*
Copyright (C) 2014-16 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnpooling.hpp"
#include "impl/pooling.hpp"

#if ENABLE_GPU
#include "datacu.hpp"
#endif

#if ENABLE_CUDNN
#include "impl/nnpooling_cudnn.hpp"
#endif
#include <assert.h>

using namespace vl ;

/* ---------------------------------------------------------------- */
/*                                                nnpooling_forward */
/* ---------------------------------------------------------------- */

#define DISPATCH(deviceType, op, type) \
status = vl::impl::op<deviceType, type>::forward \
((type*)output.getMemory(), (type const*)data.getMemory(), \
data.getHeight(), data.getWidth(), data.getDepth() * data.getSize(), \
poolHeight, poolWidth, \
strideY, strideX, \
padTop, padBottom, \
padLeft, padRight) ;

#define DISPATCH2(deviceType, op) \
switch (dataType) { \
case vlTypeFloat : DISPATCH(deviceType, op, float) ; break ; \
IF_DOUBLE(case vlTypeDouble : DISPATCH(deviceType, op, double) ; break ;) \
default: assert(false) ; return vlErrorUnknown ; \
}

#define DISPATCH3(deviceType) \
switch (method) { \
case vlPoolingAverage : DISPATCH2(deviceType, pooling_average) ; break ; \
case vlPoolingMax : DISPATCH2(deviceType, pooling_max) ; break ; \
default: assert(false) ; return vlErrorUnknown ; \
}

#define DISPATCHCUDNN(dataType) \
status = vl::impl::nnpooling_cudnn<dataType>::forward \
(context, output, data, \
method, \
poolHeight, poolWidth, \
strideY, strideX, \
padTop, padBottom, \
padLeft, padRight) ;

#define DISPATCHCUDNN2() \
switch (dataType) { \
case vlTypeFloat : DISPATCHCUDNN(vlTypeFloat) ; break ; \
IF_DOUBLE(case vlTypeDouble : DISPATCHCUDNN(vlTypeDouble) ; break ;) \
default: assert(false) ; return vlErrorUnknown ; \
}

vl::Error
vl::nnpooling_forward(vl::Context& context,
                      vl::Tensor output,
                      vl::Tensor data,
                      PoolingMethod method,
                      int poolHeight, int poolWidth,
                      int strideY, int strideX,
                      int padTop, int padBottom,
                      int padLeft, int padRight)
{
  vl::Error status = vlSuccess ;
  vl::Device deviceType = output.getDeviceType() ;
  vl::Type dataType = output.getDataType() ;

  switch (deviceType) {
    default:
      assert(false) ;
      return vl::vlErrorUnknown ;

    case vl::CPU:
      DISPATCH3(vl::CPU) ;
      break ;

#ifdef ENABLE_GPU
    case vl::GPU:
#if ENABLE_CUDNN
      if (context.getCudaHelper().getCudnnEnabled()) {
        DISPATCHCUDNN2() ;
        if (status == vl::vlSuccess) { return status ; }
        if (status != vl::vlErrorUnsupported) { return status ; }
        /* this case was not supported by CUDNN -- fallback */
      }
#endif
      DISPATCH3(GPU) ;
      if (status == vlErrorCuda) {
        context.setError(context.getCudaHelper().catchCudaError(__func__)) ;
      }
      break ;
#endif
  }
  return context.passError(status, "nnpooling_forward") ;
}

/* ---------------------------------------------------------------- */
/*                                               nnpooling_backward */
/* ---------------------------------------------------------------- */

#undef DISPATCH
#undef DISPATCH2

// backward max and average want slightly differet argument lists

#define DISPATCH_pooling_average(deviceType, type) \
status = vl::impl::pooling_average<deviceType, type>::backward \
((type*)derData.getMemory(), (type const*)derOutput.getMemory(), \
derData.getHeight(), derData.getWidth(), derData.getDepth() * derData.getSize(), \
poolHeight, poolWidth, \
strideY, strideX, \
padTop, padBottom, \
padLeft, padRight) ;

#define DISPATCH_pooling_max(deviceType, type) \
status = vl::impl::pooling_max<deviceType, type>::backward \
((type*)derData.getMemory(), (type const*)data.getMemory(), (type const*)derOutput.getMemory(), \
derData.getHeight(), derData.getWidth(), derData.getDepth() * derData.getSize(), \
poolHeight, poolWidth, \
strideY, strideX, \
padTop, padBottom, \
padLeft, padRight) ;

#define DISPATCH2(deviceType, op) \
switch (dataType) { \
case vlTypeFloat : DISPATCH_ ## op (deviceType, float) ; break ; \
IF_DOUBLE(case vlTypeDouble : DISPATCH_ ## op (deviceType, double) ; break ;) \
default: assert(false) ; return vlErrorUnknown ; \
}

vl::Error
vl::nnpooling_backward(Context& context,
                       Tensor derData,
                       Tensor data,
                       Tensor derOutput,
                       PoolingMethod method,
                       int poolHeight, int poolWidth,
                       int strideY, int strideX,
                       int padTop, int padBottom,
                       int padLeft, int padRight)
{
  vl::Error status = vlSuccess ;
  vl::Device deviceType = derOutput.getDeviceType() ;
  vl::Type dataType = derOutput.getDataType() ;

  switch (deviceType) {
    default:
      assert(false) ;
      return vl::vlErrorUnknown ;

    case vl::CPU:
      DISPATCH3(vl::CPU) ;
      break ;

#if ENABLE_GPU
    case vl::GPU:
#if ENABLE_CUDNN
      if (context.getCudaHelper().getCudnnEnabled()) {
        /*
         Unfortunately CuDNN requires both the input and the output pooling arrays
         to be available for computing derivatives, whereas MatConvNet only requires the input one.
         */
      }
#endif
      DISPATCH3(vl::GPU) ;
      if (status == vlErrorCuda) {
        context.setError(context.getCudaHelper().catchCudaError("pooling_*::backward")) ;
      }
      break ;
#endif
  }
  return context.passError(status, "nnpooling_backward") ;
}
