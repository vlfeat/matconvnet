// @file nnconv.cu
// @brief Convolution block MEX wrapper
// @author Andrea Vedaldi
// @author Max Jaderberg

/*
Copyright (C) 2014 Andrea Vedaldi and Max Jaderberg
Copyright (C) 2015 Andrea Vedaldi.

All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "bits/mexutils.h"
#include "bits/datamex.hpp"
#include "bits/nnconv.hpp"
#include "bits/nnfullyconnected.hpp"
#include "bits/nnsubsample.hpp"

#if ENABLE_GPU
#include "bits/datacu.hpp"
#endif

#include <memory>
#include <assert.h>
#include <math.h>

/* option codes */
enum {
  opt_stride = 0,
  opt_pad,
  opt_dilate,
  opt_verbose,
  opt_no_der_data,
  opt_no_der_filters,
  opt_no_der_biases,
  opt_cudnn,
  opt_no_cudnn,
  opt_cudnn_workspace_limit,
  opt_transpose
} ;

/* options */
VLMXOption  options [] = {
  {"Stride",                1,   opt_stride                },
  {"Pad",                   1,   opt_pad                   },
  {"Dilate",                1,   opt_dilate                },
  {"Verbose",               0,   opt_verbose               },
  {"NoDerData",             0,   opt_no_der_data           },
  {"NoDerFilters",          0,   opt_no_der_filters        },
  {"NoderBiases",           0,   opt_no_der_biases         },
  {"Cudnn",                 0,   opt_cudnn                 },
  {"NoCudnn",               0,   opt_no_cudnn              },
  {"CudnnWorkSpaceLimit",   1,   opt_cudnn_workspace_limit },
  {0,                       0,   0                         }
} ;

/* ---------------------------------------------------------------- */
/*                                                          Context */
/* ---------------------------------------------------------------- */

vl::MexContext context ;

/*
 Resetting the context here resolves a crash when MATLAB quits and
 the ~Context function is implicitly called on unloading the MEX file.
 */
void atExit()
{
  context.clear() ;
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
  IN_DATA = 0, IN_FILTERS, IN_BIASES, IN_DEROUTPUT, IN_END
} ;

enum {
  OUT_RESULT = 0, OUT_DERFILTERS, OUT_DERBIASES, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
  int strideX = 1 ;
  int strideY = 1 ;
  int padLeft = 0 ;
  int padRight = 0 ;
  int padTop = 0 ;
  int padBottom = 0 ;
  int dilateY = 1 ;
  int dilateX = 1 ;
  int numFilterGroups = 1 ;

  bool backMode = false ;
  bool hasFilters = false ;
  bool hasBiases = false ;
  bool fullyConnectedMode = false ;
  bool computeDerData = true ;
  bool computeDerFilters = true ;
  bool computederBiases = true ;

  int verbosity = 0 ;
  int opt ;
  int next = IN_END ;
  mxArray const *optarg ;

  /* -------------------------------------------------------------- */
  /*                                            Check the arguments */
  /* -------------------------------------------------------------- */

  mexAtExit(atExit) ;

  if (nin < 3) {
    vlmxError(VLMXE_IllegalArgument, "There are less than three arguments.") ;
  }

  if (nin > 3 && vlmxIsString(in[3],-1)) {
    next = 3 ;
    backMode = 0 ;
  } else {
    backMode = (nin >= 4) ;
  }

  while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
    switch (opt) {
      case opt_verbose :
        ++ verbosity ;
        break ;

      case opt_stride :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          vlmxError(VLMXE_IllegalArgument, "STRIDE is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            strideY = (int)mxGetPr(optarg)[0] ;
            strideX = strideY ;
            break ;
          case 2:
            strideY = (int)mxGetPr(optarg)[0] ;
            strideX = (int)mxGetPr(optarg)[1] ;
            break ;
          default:
            vlmxError(VLMXE_IllegalArgument, "STRIDE has neither one nor two elements.") ;
        }
        break ;

      case opt_pad :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          vlmxError(VLMXE_IllegalArgument, "PAD is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            padLeft = (int)mxGetPr(optarg)[0] ;
            padRight = padLeft ;
            padTop = padLeft ;
            padBottom = padLeft ;
            break ;
          case 4:
            padTop = (int)mxGetPr(optarg)[0] ;
            padBottom = (int)mxGetPr(optarg)[1] ;
            padLeft = (int)mxGetPr(optarg)[2] ;
            padRight = (int)mxGetPr(optarg)[3] ;
            break ;
          default:
            vlmxError(VLMXE_IllegalArgument, "PAD has neither one nor four elements.") ;
        }
        break ;

      case opt_dilate :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          vlmxError(VLMXE_IllegalArgument, "DILATE is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            dilateY = (int)mxGetPr(optarg)[0] ;
            dilateX = dilateY ;
            break ;
          case 2:
            dilateY = (int)mxGetPr(optarg)[0] ;
            dilateX = (int)mxGetPr(optarg)[1] ;
            break ;
          default:
            vlmxError(VLMXE_IllegalArgument, "DILATE has neither one nor two elements.") ;
        }
        break ;

      case opt_no_der_data :
        computeDerData = false ;
        break ;

      case opt_no_der_filters :
        computeDerFilters = false ;
        break ;

      case opt_no_der_biases :
        computederBiases = false ;
        break ;

      case opt_no_cudnn :
#if ENABLE_CUDNN
        context.getCudaHelper().setCudnnEnabled(false) ;
#endif
        break ;

      case opt_cudnn :
#if ENABLE_CUDNN
        context.getCudaHelper().setCudnnEnabled(true) ;
#endif
        break ;

      case opt_cudnn_workspace_limit :
      {
#if ENABLE_CUDNN
        double x ;
        if (!vlmxIsScalar(optarg) || (x = mxGetScalar(optarg)) < 0) {
          vlmxError(VLMXE_IllegalArgument, "CudnnWorkSpaceLimit is not a non-negative scalar.") ;
        }
        context.getCudaHelper().setCudnnConvolutionFwdPreference
        ((x==mxGetInf() ?
          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST :
          HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT),
         (size_t)x) ;
        context.getCudaHelper().setCudnnConvolutionBwdFilterPreference
        ((x==mxGetInf() ?
          HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST :
          HIPDNN_CONVOLUTION_BWD_FILTER_SPECIFY_WORKSPACE_LIMIT),
         (size_t)x) ;
        context.getCudaHelper().setCudnnConvolutionBwdDataPreference
        ((x==mxGetInf() ?
          HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST :
          HIPDNN_CONVOLUTION_BWD_DATA_SPECIFY_WORKSPACE_LIMIT),
         (size_t)x) ;
        break ;
#endif
      }

      default: break ;
    }
  }

  vl::MexTensor data(context) ;
  vl::MexTensor filters(context) ;
  vl::MexTensor biases(context) ;
  vl::MexTensor derOutput(context) ;

  data.init(in[IN_DATA]) ;
  data.reshape(4) ;

  filters.init(in[IN_FILTERS]) ;
  filters.reshape(4) ;

  biases.init(in[IN_BIASES]) ;

  if (backMode) {
    derOutput.init(in[IN_DEROUTPUT]) ;
    derOutput.reshape(4) ;
  }

  hasFilters = !filters.isEmpty() ;
  hasBiases = !biases.isEmpty() ;

  /* check for GPU/data class consistency */
  if (hasFilters && ! vl::areCompatible(data, filters)) {
    vlmxError(VLMXE_IllegalArgument, "DATA and FILTERS do not have compatible formats.") ;
  }
  if (hasBiases && ! vl::areCompatible(data, biases)) {
    vlmxError(VLMXE_IllegalArgument, "DATA and BIASES do not have compatible formats.") ;
  }
  if (backMode && ! vl::areCompatible(data, derOutput)) {
    vlmxError(VLMXE_IllegalArgument, "DATA and DEROUTPUT do not have compatible formats.") ;
  }

  /* basic argument checks */
  if (strideX < 1 || strideY < 1) {
    vlmxError(VLMXE_IllegalArgument, "At least one element of STRIDE is smaller than one.") ;
  }
  if (padLeft < 0 ||
      padRight < 0 ||
      padTop < 0 ||
      padBottom < 0) {
    vlmxError(VLMXE_IllegalArgument, "An element of PAD is negative.") ;
  }
  if (dilateY < 1 || dilateX < 1) {
    vlmxError(VLMXE_IllegalArgument, "An element of DILATE is less than one.") ;
  }
  if (!hasFilters && (dilateY != 1 || dilateX != 1)) {
    vlmxError(VLMXE_IllegalArgument, "There are no filters and DILATE is not one.") ;
  }

  /* Get the filter shape */
  vl::TensorShape filtersShape(filters) ;
  int equivalentNumFilters ;
  if (hasFilters) {
    if (filtersShape.getHeight() == 0 || filtersShape.getWidth() == 0 || filtersShape.getDepth() == 0) {
      vlmxError(VLMXE_IllegalArgument, "A dimension of FILTERS is void.") ;
    }
    if (data.getHeight() + (padTop+padBottom) < (filters.getHeight() - 1)*dilateY + 1 ||
        data.getWidth() + (padLeft+padRight) < (filters.getWidth() - 1)*dilateX + 1) {
      vlmxError(VLMXE_IllegalArgument, "FILTERS are larger than the DATA (including padding).") ;
    }
    /* grouped filters */
    numFilterGroups = data.getDepth() / filters.getDepth() ;
    if (numFilterGroups * filters.getDepth() != data.getDepth()) {
      vlmxError(VLMXE_IllegalArgument, "The FILTERS depth does not divide the DATA depth.") ;
    }
    if (filters.getSize() % numFilterGroups != 0) {
      vlmxError(VLMXE_IllegalArgument, "The number of filter groups does not divide the number of filters.") ;
    }
    equivalentNumFilters = filters.getSize() ;
  } else {
    /* empty filters -> pretend the identity filter bank */
    filtersShape = vl::TensorShape(1, 1, data.getDepth(), data.getDepth()) ;
    numFilterGroups = 1 ;
    equivalentNumFilters = data.getDepth() ;
  }

  /* Get the output shape */
  int kernelExtentX = (filtersShape.getWidth() - 1)*dilateX + 1 ;
  int kernelExtentY = (filtersShape.getHeight() - 1)*dilateY + 1 ;

  vl::TensorShape outputShape((data.getHeight() + (padTop+padBottom) - kernelExtentY)/strideY + 1,
                                (data.getWidth()  + (padLeft+padRight) - kernelExtentX)/strideX + 1,
                                equivalentNumFilters,
                                data.getSize()) ;

  if (backMode && (derOutput != outputShape)) {
    vlmxError(VLMXE_IllegalArgument, "DEROUTPUT dimensions are incompatible with X and FILTERS.") ;
  }

  /* Check the biases sizes */
  if (hasBiases) {
    if (biases.getNumElements() != filtersShape.getSize()) {
      vlmxError(VLMXE_IllegalArgument, "The number of elements of BIASES is not the same as the number of filters.") ;
    }
  }

  /*
   Detect fully connected mode (further optimisations):
   the output is 1 x 1 pixels,
   no padding,
   one filter group,
   stride of one pixel
   */
  fullyConnectedMode = (outputShape.getHeight() == 1 &&
                        outputShape.getWidth() == 1 &&
                        strideY == 1 &&
                        strideX == 1 &&
                        padTop == 0 &&
                        padBottom == 0 &&
                        padLeft == 0 &&
                        padRight == 0 &&
                        dilateY == 1 &&
                        dilateX == 1 &&
                        numFilterGroups == 1) ;

  /* create output buffers */
  vl::DeviceType deviceType = data.getDeviceType() ;
  vl::DataType dataType = data.getDataType() ;
  vl::MexTensor output(context) ;
  vl::MexTensor derData(context) ;
  vl::MexTensor derFilters(context) ;
  vl::MexTensor derBiases(context) ;

  if (!backMode) {
    output.init(deviceType, dataType, outputShape) ;
  } else {
    if (computeDerData) {
      derData.init(deviceType, dataType, data.getShape()) ;
    }
    if (computeDerFilters && hasFilters) {
      derFilters.init(deviceType, dataType, filters.getShape()) ;
    }
    if (computederBiases && hasBiases) {
      derBiases.init(deviceType, dataType, biases.getShape()) ;
    }
  }

  if (verbosity > 0) {
    mexPrintf("vl_nnconv: %s; %s", backMode?"backward":"forward", (data.getDeviceType()==vl::VLDT_GPU) ? "GPU" : "CPU") ;
    if (data.getDeviceType() == vl::VLDT_GPU) {
#if ENABLE_CUDNN
      mexPrintf("; %s\n", context.getCudaHelper().getCudnnEnabled() ? "cuDNN" : "cuBLAS") ;
#else
      mexPrintf("; cuBLAS\n") ;
#endif
    } else {
      mexPrintf("; BLAS\n") ;
    }
    mexPrintf("vl_nnconv: stride: [%d %d], pad: [%d %d %d %d], dilate: [%d %d]\n"
              "vl_nnconv: num filter groups: %d, has bias: %d, has filters: %d, is fully connected: %d\n",
              strideY, strideX,
              padTop, padBottom, padLeft, padRight,
              dilateY, dilateX,
              numFilterGroups, hasBiases, hasFilters, fullyConnectedMode) ;
    vl::print("vl_nnconv: data: ", data) ;
    if (hasFilters) { vl::print("vl_nnconv: filters: ", filters) ; }
    if (hasBiases) { vl::print("vl_nnconv: biases: ", biases) ; }
    if (backMode) {
      vl::print("vl_nnconv: derOutput: ", derOutput) ;
      vl::print("vl_nnconv: derData: ", derData) ;
      if (hasFilters) { vl::print("vl_nnconv: derFilters: ", derFilters) ; }
      if (hasBiases) { vl::print("vl_nnconv: derBiases: ", derBiases) ; }
    } else {
      vl::print("vl_nnconv: output: ", output) ;
    }
  }

  /* -------------------------------------------------------------- */
  /*                                                    Do the work */
  /* -------------------------------------------------------------- */

  vl::ErrorCode error ;

  /*
   special case: fully connected
   (could be done as a regular case, but it is faster this way)
   */
  if (fullyConnectedMode) {
    vl::nn::FullyConnected op(context) ;
    if (!backMode) {
      error = op.forward(output,data,filters,biases) ;
    } else {
      error = op.backward(derData,derFilters,derBiases,data,filters,derOutput) ;
    }
    goto doneok ;
  }

  /* special case: no filters = identity filter bank (subsample + bias) */
  if (!hasFilters) {
    vl::nn::Subsample op(context,strideY,strideX,
                         padTop,padBottom,padLeft,padRight) ;
    if (!backMode) {
      error = op.forwardWithBias(output,data,biases) ;
    } else {
      error = op.backwardWithBias(derData,derBiases,derOutput) ;
    }
    goto doneok ;
  }

  /* regular case */
  {
    vl::nn::Convolution op(context,
                           strideY, strideX,
                           padTop, padBottom, padLeft, padRight,
                           dilateY, dilateX);
    if (!backMode) {
      error = op.forward(output, 0,
                         data, 1,
                         filters,
                         biases) ;
    } else {
      error = op.backward(derData,
                          derFilters,
                          derBiases,
                          data,
                          filters,
                          derOutput) ;
    }
  }

doneok:
  if (verbosity > 0) {
#if ENABLE_CUDNN
    if (context.getCudaHelper().getCudnnEnabled()) {
      mexPrintf("vl_nnconv: cuDNN workspace used: "
                "fwd %.6g MB"
                ", bwd filter %.6g MB"
                ", bwd data %.6g MB\n",
                (double)context.getCudaHelper().getCudnnConvolutionFwdWorkSpaceUsed() / (1024*1024),
                (double)context.getCudaHelper().getCudnnConvolutionBwdFilterWorkSpaceUsed() / (1024*1024),
                (double)context.getCudaHelper().getCudnnConvolutionBwdDataWorkSpaceUsed() / (1024*1024)) ;
    }
#endif
  }

  /* -------------------------------------------------------------- */
  /*                                                        Cleanup */
  /* -------------------------------------------------------------- */

  if (error != vl::VLE_Success) {
    vlmxError(VLMXE_IllegalArgument, context.getLastErrorMessage().c_str()) ;
  }
  if (backMode) {
    mxClassID classID ;
    switch (derOutput.getDataType()) {
      case vl::VLDT_Float: classID = mxSINGLE_CLASS ; break ;
      case vl::VLDT_Double: classID = mxDOUBLE_CLASS ; break ;
      default: abort() ;
    }
    out[OUT_RESULT] = (computeDerData) ? derData.relinquish() : mxCreateNumericMatrix(0,0,classID,mxREAL) ;
    out[OUT_DERFILTERS] = (computeDerFilters & hasFilters)? derFilters.relinquish() : mxCreateNumericMatrix(0,0,classID,mxREAL) ;
    out[OUT_DERBIASES] = (computederBiases & hasBiases) ? derBiases.relinquish() : mxCreateNumericMatrix(0,0,classID,mxREAL) ;
  } else {
    out[OUT_RESULT] = output.relinquish() ;
  }
}
